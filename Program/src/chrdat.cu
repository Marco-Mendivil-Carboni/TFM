#include "hip/hip_runtime.h"
//Includes

#include "chrdat.cuh" //chromatin data

//Namespace

namespace mmc //Marco Mendívil Carboni
{

//Functions

//chromatin data constructor
chrdat::chrdat(parmap &par) //parameters
  : N {par.get_val<uint>("number_of_particles",0)}
  , R {par.get_val<float>("confinement_radius",-1.0)}
  , T {par.get_val<float>("temperature",298.0)}
  , n_l {par.get_val<uint>("number_of_lbs",0)}
  , i_f {0}, t {0.0}
{
  //check parameters
  if (!(1<=N&&N<100'000)){ throw error("number_of_particles out of range");}
  if (!(0.0<R&&R<100.0)){ throw error("confinement_radius out of range");}
  if (!(0.0<T&&T<1'000.0)){ throw error("temperature out of range");}
  if (!(n_l<100'000)){ throw error("number_of_lbs out of range");}
  float cvf = N*pow(0.5/(R-0.5),3.0); //chromatin volume fraction
  if (cvf>0.5){ throw error("chromatin volume fraction above 0.5");}
  float laf = n_l*pow(lco/(R-rco),2.0); //lbs area fraction
  if (laf>0.6){ throw error("lbs area fraction above 0.6");}
  std::string msg_1 = ""; //1st message
  msg_1 += "N = "+cnfs(N,5,'0')+" ";
  msg_1 += "R = "+cnfs(R,5,'0',2)+" ";
  msg_1 += "T = "+cnfs(T,5,'0',1)+" ";
  logger::record(msg_1);
  std::string msg_2 = ""; //2nd message
  msg_2 += "n_l = "+cnfs(n_l,5,'0')+" ";
  msg_2 += "cvf = "+cnfs(cvf,5,'0',3)+" ";
  msg_2 += "laf = "+cnfs(laf,5,'0',3)+" ";
  logger::record(msg_2);

  //allocate device memory
  cuda_check(hipMalloc(&pt,N*sizeof(ptype)));
  cuda_check(hipMalloc(&r,N*sizeof(vec3f)));
  cuda_check(hipMalloc(&f,N*sizeof(vec3f)));
  cuda_check(hipMalloc(&lr,n_l*sizeof(vec3f)));

  //allocate host memory
  cuda_check(hipHostMalloc(&hpt,N*sizeof(ptype)));
  cuda_check(hipHostMalloc(&hr,N*sizeof(vec3f)));
  cuda_check(hipHostMalloc(&hf,N*sizeof(vec3f)));
  cuda_check(hipHostMalloc(&hlr,n_l*sizeof(vec3f)));
}

//chromatin data destructor
chrdat::~chrdat()
{
  //deallocate device memory
  cuda_check(hipFree(pt));
  cuda_check(hipFree(r));
  cuda_check(hipFree(f));
  cuda_check(hipFree(lr));

  //deallocate host memory
  cuda_check(hipHostFree(hpt));
  cuda_check(hipHostFree(hr));
  cuda_check(hipHostFree(hf));
  cuda_check(hipHostFree(hlr));
}

//write frame to text file
void chrdat::write_frame_txt(std::ofstream &txt_out_f) //text output file
{
  //write chromatin data
  txt_out_f<<i_f<<" "<<t<<"\n";
  txt_out_f<<cnfs(N,5,' ')<<"\n";
  char ptc; //particle type character
  for (uint i_p = 0; i_p<N; ++i_p) //particle index
  {
    ptc = (hpt[i_p]==LND)?'A':'B';
    txt_out_f<<std::setw(5)<<i_p+1<<std::left<<std::setw(5)<<ptc;
    txt_out_f<<std::right<<std::setw(5)<<ptc<<std::setw(5)<<i_p+1;
    txt_out_f<<cnfs(hr[i_p].x,8,' ',3);
    txt_out_f<<cnfs(hr[i_p].y,8,' ',3);
    txt_out_f<<cnfs(hr[i_p].z,8,' ',3);
    txt_out_f<<"\n";
  }
  txt_out_f<<cnfs(0.0,10,' ',5);
  txt_out_f<<cnfs(0.0,10,' ',5);
  txt_out_f<<cnfs(0.0,10,' ',5);
  txt_out_f<<"\n";

  //check filestream
  if (txt_out_f.fail())
  {
    throw mmc::error("failed to write frame to text file");
  }
}

//read frame from text file
void chrdat::read_frame_txt(std::ifstream &txt_inp_f) //text input file
{
  //read chromatin data
  std::string aux_str; //auxiliary string
  txt_inp_f>>i_f>>t;
  txt_inp_f>>aux_str;
  char ptc; //particle type character
  for (uint i_p = 0; i_p<N; ++i_p) //particle index
  {
    txt_inp_f>>aux_str;
    txt_inp_f>>ptc>>aux_str;
    hpt[i_p] = (ptc=='A')?LND:LAD;
    txt_inp_f>>hr[i_p].x;
    txt_inp_f>>hr[i_p].y;
    txt_inp_f>>hr[i_p].z;
  }
  txt_inp_f>>aux_str;
  txt_inp_f>>aux_str;
  txt_inp_f>>aux_str;

  //copy host arrays to device
  cuda_check(hipMemcpy(pt,hpt,N*sizeof(ptype),hipMemcpyHostToDevice));
  cuda_check(hipMemcpy(r,hr,N*sizeof(vec3f),hipMemcpyHostToDevice));

  //check filestream
  if (txt_inp_f.fail())
  {
    throw mmc::error("failed to read frame from text file");
  }
}

//write frame to binary file
void chrdat::write_frame_bin(std::ofstream &bin_out_f) //binary output file
{
  //------------------------------note------------------------------
  //this is a minimal trr file writing routine that doesn't rely on \ 
  //the xdr library but only works with vmd in little endian systems

  //frame header, for more information on its contents see chemfiles
  uint32_t header[18] = {1993, 1, 0, 
    0, 0, 0, 0, 0, 0, 0, 3*N*4, 0, 0, N, i_f, 0, 
    reinterpret_cast<uint32_t &>(t), 0};

  //write chromatin data
  bin_out_f.write(reinterpret_cast<char *>(header),sizeof(header));
  for (uint i_p = 0; i_p<N; ++i_p) //particle index
  {
    bin_out_f.write(reinterpret_cast<char *>(&(hr[i_p].x)),4);
    bin_out_f.write(reinterpret_cast<char *>(&(hr[i_p].y)),4);
    bin_out_f.write(reinterpret_cast<char *>(&(hr[i_p].z)),4);
  }

  //check filestream
  if (bin_out_f.fail())
  {
    throw mmc::error("failed to write frame to binary file");
  }
}

//read frame from binary file
void chrdat::read_frame_bin(std::ifstream &bin_inp_f) //binary input file
{
  //read chromatin data
  uint32_t header[18]; //frame header
  bin_inp_f.read(reinterpret_cast<char *>(header),sizeof(header));
  i_f = header[14]; t = reinterpret_cast<float &>(header[16]);
  for (uint i_p = 0; i_p<N; ++i_p) //particle index
  {
    bin_inp_f.read(reinterpret_cast<char *>(&(hr[i_p].x)),4);
    bin_inp_f.read(reinterpret_cast<char *>(&(hr[i_p].y)),4);
    bin_inp_f.read(reinterpret_cast<char *>(&(hr[i_p].z)),4);
  }

  //copy host position array to device
  cuda_check(hipMemcpy(r,hr,N*sizeof(vec3f),hipMemcpyHostToDevice));

  //check filestream
  if (bin_inp_f.fail())
  {
    throw mmc::error("failed to read frame from binary file");
  }
}

} //namespace mmc
