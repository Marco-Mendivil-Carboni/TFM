#include "hip/hip_runtime.h"
//Includes

#include "chrsim.cuh" //chromatin simulation

#include <time.h> //time utilities library

#include <hiprand/hiprand_kernel.h> //cuRAND device functions

//Namespace

namespace mmc //Marco Mendívil Carboni
{

//Constants

static constexpr float dt = 1.0/2048; //timestep
static constexpr float rco = 1.154701; //Wang-Frenkel repulsive cutoff
static constexpr float aco = 2.000000; //Wang-Frenkel attractive cutoff

//Aliases

using prng = hiprandStatePhilox4_32_10; //PRNG type

//Enumerations

enum srint //short-range interaction
{
  WFI, //Wang-Frenkel interaction
  SRI //Soft-Repulsive interaction
};

//Device Functions

//calculate bonded forces
inline __device__ void calc_bf(
  const uint N, //number of particles
  uint i_p, //particle index
  float4 *r, //position array
  float4 *f) //force array
{
  //declare auxiliary variables
  float3 vec[4]; //bond vectors
  float il[4]; //bond inverse lengths
  float cos[3]; //bond angle cosines
  float3 bf = {0.0,0.0,0.0}; //bonded forces

  //calculate bond vectors, inverse lengths and angle cosines
  for (uint i_b = 0; i_b<4; ++i_b) //bond index
  {
    if ((i_p+i_b)>=2 && (i_p+i_b)<=N) //calculate variables if bond exists
    {
      vec[i_b] = make_float3(r[i_p+i_b-1]-r[i_p+i_b-2]);
      il[i_b] = rsqrtf(dot(vec[i_b],vec[i_b]));
    }
    else //set variables to zero if bond doesn't exist
    {
      vec[i_b] = {0.0,0.0,0.0};
      il[i_b] = 0.0;
    }
  }
  for (uint i_c = 0; i_c<3; ++i_c) //cosine index
  {
    cos[i_c] = dot(vec[i_c+1],vec[i_c])*il[i_c+1]*il[i_c];
  }

  //calculate elastic potential force
  bf += k_e*(+(1.0-l_0*il[2])*vec[2]-(1.0-l_0*il[1])*vec[1]);

  //calculate bending potential force
  bf += k_b*(+il[1]*il[0]*vec[0]-cos[0]*vec[1]*il[1]*il[1]);
  bf += k_b*(+il[1]*il[2]*vec[2]-cos[1]*vec[1]*il[1]*il[1]);
  bf += k_b*(-il[2]*il[1]*vec[1]+cos[1]*vec[2]*il[2]*il[2]);
  bf += k_b*(-il[2]*il[3]*vec[3]+cos[2]*vec[2]*il[2]*il[2]);

  //add result to force array
  f[i_p] += make_float4(bf);
}

//calculate confinement force
inline __device__ void calc_cf(
  const float R, //confinement radius
  uint i_p, //particle index
  float4 *r, //position array
  float4 *f) //force array
{
  //calculate auxiliary variables
  float d_r; //radial distance to origin
  d_r = length(make_float3(r[i_p]));
  float dwp = (R+0.5)-d_r; //wall-particle distance
  if (dwp>rco){ return;}

  //calculate confinement force
  float3 cf = make_float3(-r[i_p]/d_r); //confinement force
  float d2 = dwp*dwp; //dwp squared
  cf *= (18*d2*d2-96*d2+96)/(d2*d2*d2*d2);

  //add result to force array
  f[i_p] += make_float4(cf);
}

//calculate short-range force
template <srint I> inline __device__ void calc_srf(
  const float eps, //particle energy
  float3 vpp, //particle particle vector
  float dpp, //particle particle distance
  float3 &srf); //short-range forces

//calculate Wang-Frenkel force
template <> inline __device__ void calc_srf<WFI>(
  const float eps, //particle energy
  float3 vpp, //particle particle vector
  float dpp, //particle particle distance
  float3 &srf) //short-range forces
{
  float d2 = dpp*dpp; //dpp squared
  srf += eps*(18*d2*d2-96*d2+96)/(d2*d2*d2*d2)*vpp;
}

//calculate Soft-Repulsive force
template <> inline __device__ void calc_srf<SRI>(
  const float eps, //particle energy
  float3 vpp, //particle particle vector
  float dpp, //particle particle distance
  float3 &srf) //short-range forces
{
  srf += (12-6*dpp)*vpp;
}

//calculate short-range forces with cell's particles
template <srint I> inline __device__ void calc_cell_srf(
  const float eps, //particle energy
  uint i_c, //cell index
  uint i_p, //particle index
  float3 r_i, //particle position
  float4 *r, //position array
  sugrid *srg_p, //short-range grid pointer
  float3 &srf) //short-range forces
{
  //declare auxiliary variables
  uint j_p; //secondary particle index
  float3 r_j; //secondary particle position
  uint beg = srg_p->beg[i_c]; //cell beginning
  uint end = srg_p->end[i_c]; //cell end

  //check cell isn't empty
  if (beg==0xffffffff){ return;}

  //range over cell's particles
  for (uint sai = beg; sai<end; ++sai) //sorted array index
  {
    //get secondary particle index
    j_p = srg_p->spi[sai];

    //calculate force only between non-bonded particles
    if (((j_p>i_p)?j_p-i_p:i_p-j_p)>1)
    {
      //calculate particle particle distance
      r_j = make_float3(r[j_p]);
      float3 vpp = r_i-r_j; //particle particle vector
      float dpp = length(vpp); //particle particle distance
      if (dpp>aco){ continue;}

      //calculate short-range force
      calc_srf<I>(eps,vpp,dpp,srf);
    }
  }
}

//calculate all short-range forces
template <srint I> inline __device__ void calc_all_srf(
  const float eps, //particle energy
  uint i_p, //particle index
  float4 *r, //position array
  float4 *f, //force array
  sugrid *srg_p) //short-range grid pointer
{
  //calculate auxiliary variables
  float3 r_i = make_float3(r[i_p]); //particle position
  const float csl = srg_p->csl; //grid cell side length
  const uint cps = srg_p->cps; //grid cells per side
  const uint n_c = srg_p->n_c; //number of grid cells
  int3 ir = floorf(r_i/csl); //integer coordinates
  uint iofst = (cps/2)*(1+cps+cps*cps); //index offset
  float3 srf = {0.0,0.0,0.0}; //short-range forces

  //range over neighbouring cells
  uint nci; //neighbour cell index
  int3 nir; //neighbour integer coordinates
  for (nir.x = ir.x-1; nir.x<=ir.x+1; ++nir.x)
  {
    for (nir.y = ir.y-1; nir.y<=ir.y+1; ++nir.y)
    {
      for (nir.z = ir.z-1; nir.z<=ir.z+1; ++nir.z)
      {
        //calculate neighbour cell index
        nci = iofst+nir.x+nir.y*cps+nir.z*cps*cps;
        if (nci>=n_c){ continue;}

        //calculate short-range forces with cell's particles
        calc_cell_srf<I>(eps,nci,i_p,r_i,r,srg_p,srf);
      }
    }
  }

  //add result to force array
  f[i_p] += make_float4(srf);
}

//Global Functions

//initialize PRNG state array
__global__ void init_ps(
  const uint N, //number of particles
  void *vps, //void PRNG state array
  uint pseed) //PRNG seed
{
  //calculate particle index
  uint i_p = blockIdx.x*blockDim.x+threadIdx.x; //particle index
  if (i_p>=N){ return;}

  //initialize PRNG state
  prng *ps = static_cast<prng *>(vps); //PRNG state array
  hiprand_init(pseed,i_p,0,&ps[i_p]);
}

//execute 1st stage of the Runge-Kutta method
template <srint I> __global__ void exec_RK_1(
  const uint N, //number of particles
  const float R, //confinement radius
  const float eps, //particle energy
  float4 *r, //position array
  float4 *f, //force array
  float4 *er, //extra position array
  float sd, //standard deviation
  float4 *rn, //random number array
  void *vps, //void PRNG state array
  sugrid *srg_p) //short-range grid pointer
{
  //calculate particle index
  uint i_p = blockIdx.x*blockDim.x+threadIdx.x; //particle index
  if (i_p>=N){ return;}

  //calculate random numbers
  float3 az; //absolute z-score
  prng *ps = static_cast<prng *>(vps); //PRNG state array
  do
  {
    rn[i_p] = sd*hiprand_normal4(&ps[i_p]);
    az = fabs(make_float3(rn[i_p])/sd);
  }
  while (az.x>5||az.y>5||az.z>5);

  //calculate forces
  f[i_p] = {0.0,0.0,0.0,0.0};
  calc_bf(N,i_p,r,f);
  calc_cf(R,i_p,r,f);
  calc_all_srf<I>(eps,i_p,r,f,srg_p);

  //calculate extra position
  er[i_p] = r[i_p]+f[i_p]*dt+rn[i_p];
}

//execute 2nd stage of the Runge-Kutta method
template <srint I> __global__ void exec_RK_2(
  const uint N, //number of particles
  const float R, //confinement radius
  const float eps, //particle energy
  float4 *r, //position array
  float4 *f, //force array
  float4 *er, //extra position array
  float4 *ef, //extra force array
  float4 *rn, //random number array
  sugrid *srg_p) //short-range grid pointer
{
  //calculate particle index
  uint i_p = blockIdx.x*blockDim.x+threadIdx.x; //particle index
  if (i_p>=N){ return;}

  //calculate forces
  ef[i_p] = {0.0,0.0,0.0,0.0};
  calc_bf(N,i_p,er,ef);
  calc_cf(R,i_p,er,ef);
  calc_all_srf<I>(eps,i_p,er,ef,srg_p);

  //calculate new position
  r[i_p] = r[i_p]+0.5*(ef[i_p]+f[i_p])*dt+rn[i_p];
}

//Host Functions

//chromatin simulation constructor
chrsim::chrsim(parmap &par) //parameters
  : chrdat(par)
  , fpf {par.get_val<uint>("frames_per_file",100)}
  , spf {par.get_val<uint>("steps_per_frame",1*2048)}
  , tpb {par.get_val<uint>("threads_per_block",256)}
  , sd {sqrtf(2.0*k_B*T*dt)}
  , srg(N,aco,2*ceil(R/aco))
{
  //check parameters
  if (!(1<=fpf&&fpf<10'000)){ throw error("frames_per_file out of range");}
  if (!(1<=spf&&spf<10'000)){ throw error("steps_per_frame out of range");}
  if (!(1<=tpb&&tpb<1'025)){ throw error("threads_per_block out of range");}
  std::string msg = "chrsim:"; //message
  msg += " fpf = "+cnfs(fpf,4,'0');
  msg += " spf = "+cnfs(spf,4,'0');
  msg += " tpb = "+cnfs(tpb,4,'0');
  logger::record(msg);

  //allocate device memory
  cuda_check(hipMalloc(&er,N*sizeof(float4)));
  cuda_check(hipMalloc(&ef,N*sizeof(float4)));
  cuda_check(hipMalloc(&rn,N*sizeof(float4)));
  cuda_check(hipMalloc(&vps,N*sizeof(prng)));
  cuda_check(hipMalloc(&srg_p,sizeof(sugrid)));

  //copy short-range grid to device
  cuda_check(hipMemcpy(srg_p,&srg,sizeof(sugrid),hipMemcpyHostToDevice));

  //initialize PRNG
  init_ps<<<(N+tpb-1)/tpb,tpb>>>(N,vps,time(nullptr));
}

//chromatin simulation destructor
chrsim::~chrsim()
{
  //deallocate device memory
  cuda_check(hipFree(er));
  cuda_check(hipFree(ef));
  cuda_check(hipFree(rn));
  cuda_check(hipFree(vps));
  cuda_check(hipFree(srg_p));
}

//generate a random initial condition
void chrsim::generate_initial_condition()
{
  //set random particle types
  set_particle_types();

  //perform a confined random walk
  perform_random_walk();

  //separate beads
  logger::record("bead separation begun");
  // while () //check beads are separated ---------------------------------------
  // {
  //   make_RK_iteration();
  // }
  cuda_check(hipMemcpy(hr,r,N*sizeof(float4),hipMemcpyDeviceToHost));
  logger::record("bead separation ended");

  //record success message
  logger::record("initial condition generated");
}

//save simulation state to binary file
void chrsim::save_checkpoint(std::ofstream &bin_out_f) //binary output file
{
  bin_out_f.write(reinterpret_cast<char *>(&i_f),sizeof(i_f));
  bin_out_f.write(reinterpret_cast<char *>(&t),sizeof(t));
  bin_out_f.write(reinterpret_cast<char *>(hr),N*sizeof(float4));
  logger::record("simulation checkpoint saved");
}

//load simulation state from binary file
void chrsim::load_checkpoint(std::ifstream &bin_inp_f) //binary input file
{
  bin_inp_f.read(reinterpret_cast<char *>(&i_f),sizeof(i_f));
  bin_inp_f.read(reinterpret_cast<char *>(&t),sizeof(t));
  bin_inp_f.read(reinterpret_cast<char *>(hr),N*sizeof(float4));
  cuda_check(hipMemcpy(r,hr,N*sizeof(float4),hipMemcpyHostToDevice));
  logger::record("simulation checkpoint loaded");
}

//run simulation and write trajectory to binary file
void chrsim::run_simulation(std::ofstream &bin_out_f) //binary output file
{
  logger::record("simulation begun");
  for (uint ffi = 0; ffi<fpf; ++ffi) //file frame index
  {
    logger::show_prog_pc(100.0*ffi/fpf);
    for (uint fsi = 0; fsi<spf; ++fsi) //frame step index
    {
      make_RK_iteration();
    }
    cuda_check(hipMemcpy(hr,r,N*sizeof(float4),hipMemcpyDeviceToHost));
    ++i_f; t += spf*dt;
    write_frame_bin(bin_out_f);
  }
  logger::record("simulation ended");
}

//set random particle types
void chrsim::set_particle_types()
{
  //initialize host PRNG
  hiprandGenerator_t gen; //host PRNG
  hiprandCreateGeneratorHost(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen,time(nullptr));

  //set particle types randomly
  float ran; //random number in (0,1]
  for (uint i_p = 0; i_p<N; ++i_p) //particle index
  {
    hiprandGenerateUniform(gen,&ran,1);
    if (ran<0.5){ hpt[i_p] = LAD;}
    else{ hpt[i_p] = LND;}
  }

  //copy host particle type array to device
  cuda_check(hipMemcpy(pt,hpt,N*sizeof(ptype),hipMemcpyHostToDevice));

  //free host PRNG
  hiprandDestroyGenerator(gen);
}

//perform a confined random walk
void chrsim::perform_random_walk()
{
  //initialize host PRNG
  hiprandGenerator_t gen; //host PRNG
  hiprandCreateGeneratorHost(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen,time(nullptr));

  //declare auxiliary variables
  float iT = 1.0/(k_B*T); //inverse temperature
  float ran; //random number in (0,1]
  float theta; //polar angle
  float phi; //azimuthal angle
  float len_b; //bond length
  float angle_b; //bond angle
  float3 old_dir; //old direction
  float3 new_dir; //new direction
  float3 ran_dir; //random direction
  float3 per_dir; //perpendicular direction

  //place first particle
  hr[0] = {0.0,0.0,0.0,0.0};
  hiprandGenerateUniform(gen,&ran,1); theta = acos(1.0-2.0*ran);
  hiprandGenerateUniform(gen,&ran,1); phi = 2.0*M_PI*ran;
  ran_dir = {sin(theta)*cos(phi),sin(theta)*sin(phi),cos(theta)};
  old_dir = ran_dir;

  //place the rest of particles
  uint att = 0; //number of attempts
  for (uint i_p = 1; i_p<N; ++i_p) //particle index
  {
    //generate random direction perpendicular to old direction
    hiprandGenerateUniform(gen,&ran,1); theta = acos(1.0-2.0*ran);
    hiprandGenerateUniform(gen,&ran,1); phi = 2.0*M_PI*ran;
    ran_dir = {sin(theta)*cos(phi),sin(theta)*sin(phi),cos(theta)};
    per_dir = cross(old_dir,ran_dir);
    per_dir = normalize(per_dir);

    //generate random bond angle and calculate new direction
    hiprandGenerateUniform(gen,&ran,1);
    angle_b = acos(1.0+log(1.0-(1.0-exp(-2.0*(k_b*iT)))*ran)/(k_b*iT));
    new_dir = cos(angle_b)*old_dir+sin(angle_b)*per_dir;

    //calculate position of next particle
    hiprandGenerateUniform(gen,&ran,1);
    len_b = l_0+sqrt(2.0/(k_e*iT))*erfinv(2.0*ran-1.0);
    hr[i_p] = make_float4(len_b*new_dir)+hr[i_p-1];

    //check if new position is acceptable
    bool p_a = true; //position is acceptable
    if (!isfinite(hr[i_p].x)){ p_a = false;}
    if (!isfinite(hr[i_p].y)){ p_a = false;}
    if (!isfinite(hr[i_p].z)){ p_a = false;}
    for (uint j_p = 0; j_p<(i_p-1); ++j_p) //secondary particle index//remove ---
    {
      float dpp; //particle-particle distance
      dpp = length(make_float3(hr[j_p]-hr[i_p]));
      if (dpp<1.0){ p_a = false; break;}
    }
    float d_r; //radial distance to origin
    d_r = length(make_float3(hr[i_p]));
    if (((R+0.5)-d_r)<1.0){ p_a = false;}

    if (p_a) //continue
    {
      old_dir = new_dir;
      att = 0;
    }
    else //go back
    {
      ++att;
      if (att>1024){ i_p = 1;}
      else{ --i_p;}
    }
  }

  //copy host position array to device
  cuda_check(hipMemcpy(r,hr,N*sizeof(float4),hipMemcpyHostToDevice));

  //free host PRNG
  hiprandDestroyGenerator(gen);
}

//make one iteration of the Runge-Kutta method
void chrsim::make_RK_iteration()
{
  srg.generate_arrays(tpb,r);
  exec_RK_1<WFI><<<(N+tpb-1)/tpb,tpb>>>(N,R,eps,r,f,er,sd,rn,vps,srg_p);
  srg.generate_arrays(tpb,er);
  exec_RK_2<WFI><<<(N+tpb-1)/tpb,tpb>>>(N,R,eps,r,f,er,ef,rn,srg_p);
}

} //namespace mmc
