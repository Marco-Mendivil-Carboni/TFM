#include "hip/hip_runtime.h"
//Includes

#include <cstdio> //standard input and output library
#include <cmath> //mathematical functions library
#include <ctime> //time utilities library

#include "../inc/utilities.cuh"
#include "../inc/chrsim.cuh"

#include <hiprand/hiprand_kernel.h> //cuRAND device functions

//Namespace

namespace mmcc //Marco Mendívil Carboni code
{

//Constants

static constexpr float xi  = 1.000000; //damping coefficient
static constexpr float k_B = 0.001120; //Boltzmann constant
static constexpr float l_0 = 1.000000; //bond natural length
static constexpr float k_e = 100.0000; //elastic constant
static constexpr float k_b = 2.000000; //bending constant
static constexpr float r_c = 1.122462; //LJ cutoff radius
static constexpr float dt  = 1.0/2048; //timestep

static constexpr int n_s = 1*2048; //MD steps between frames

//Host Functions

chrsim::chrsim(FILE *f_ptr_par)
{
  read_parameters(f_ptr_par);

  n_blocks = (ap.N+threads_block-1)/threads_block;
  n_threads = n_blocks*threads_block;

  allocate_soa3(r_2,ap.N);
  allocate_soa3(r_1,ap.N);

  allocate_soa3(f_2,ap.N);
  allocate_soa3(f_1,ap.N);

  allocate_soa3(nrn,n_threads);

  cuda_check(hipMallocManaged(&state,n_threads*sizeof(PRNGstate)));
}

chrsim::~chrsim()
{
  free_soa3(r_2);
  free_soa3(r_1);

  free_soa3(f_2);
  free_soa3(f_1);

  free_soa3(nrn);

  hipFree(state);
}

void chrsim::read_parameters(FILE *f_ptr_par)
{
  if (std::fscanf(f_ptr_par,"T\t%f\n",&(ap.T))!=1
    ||std::fscanf(f_ptr_par,"N\t%d\n",&(ap.N))!=1
    ||std::fscanf(f_ptr_par,"R\t%f\n",&(ap.R))!=1
    ||std::fscanf(f_ptr_par,"F\t%d\n",&(ap.F))!=1)
  {
    throw error("unable to read parameters");
  }
  if ((ap.T)<__FLT_MIN__){ throw error("T must be positive");}
  if ((ap.N)<__FLT_MIN__){ throw error("N must be positive");}
  if ((ap.R)<__FLT_MIN__){ throw error("R must be positive");}
  if ((ap.F)<__FLT_MIN__){ throw error("F must be positive");}
}

void chrsim::generate_initial_configuration()
{
  //initialize PRNG
  hiprandGenerator_t gen;
  hiprandCreateGeneratorHost(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen,time(nullptr));

  //declare auxiliary variables
  float beta = 1.0/(k_B*ap.T);
  float rand, theta, varphi, bondlen, bondangle, perdirlen;
  float3 olddir, newdir, perdir;

  //place first particle
  hiprandGenerateUniform(gen,&rand,1); theta = acos(1.0-2.0*rand);
  hiprandGenerateUniform(gen,&rand,1); varphi = 2.0*M_PI*rand;
  olddir.x = sin(theta)*cos(varphi);
  olddir.y = sin(theta)*sin(varphi);
  olddir.z = cos(theta);
  r_2.x[0] = r_2.y[0] = r_2.z[0] = 0.0;

  //perform random walk
  int n_failures = 0;
  for (int i_p = 1; i_p<ap.N; ++i_p)
  {
    //generate random direction perpendicular to old direction
    hiprandGenerateUniform(gen,&rand,1); theta = acos(1.0-2.0*rand);
    hiprandGenerateUniform(gen,&rand,1); varphi = 2.0*M_PI*rand;
    perdir.x = olddir.y*cos(theta)-olddir.z*sin(theta)*sin(varphi);
    perdir.y = olddir.z*sin(theta)*cos(varphi)-olddir.x*cos(theta);
    perdir.z = olddir.x*sin(theta)*sin(varphi)-olddir.y*sin(theta)*cos(varphi);
    perdirlen = sqrt(perdir.x*perdir.x+perdir.y*perdir.y+perdir.z*perdir.z);
    perdir.x /= perdirlen; perdir.y /= perdirlen; perdir.z /= perdirlen;

    //generate random bond angle and calculate new direction
    hiprandGenerateUniform(gen,&rand,1);
    bondangle = acos(1.0+log(1.0-(1.0-exp(-2.0*(k_b*beta)))*rand)/(k_b*beta));
    newdir.x = olddir.x*cos(bondangle)+perdir.x*sin(bondangle);
    newdir.y = olddir.y*cos(bondangle)+perdir.y*sin(bondangle);
    newdir.z = olddir.z*cos(bondangle)+perdir.z*sin(bondangle);

    //calculate position of next particle
    hiprandGenerateUniform(gen,&rand,1);
    bondlen = l_0+sqrt(2.0/(k_e*beta))*erfinv(2.0*rand-1.0);
    r_2.x[i_p] = bondlen*newdir.x+r_2.x[i_p-1];
    r_2.y[i_p] = bondlen*newdir.y+r_2.y[i_p-1];
    r_2.z[i_p] = bondlen*newdir.z+r_2.z[i_p-1];

    //check if new position is acceptable
    int accept = 1;
    if (!isfinite(r_2.x[i_p])){ accept = 0;}
    if (!isfinite(r_2.y[i_p])){ accept = 0;}
    if (!isfinite(r_2.z[i_p])){ accept = 0;}
    for (int j_p = 0; j_p<(i_p-1); ++j_p)
    {
      float dist = 0.0;
      dist += (r_2.x[j_p]-r_2.x[i_p])*(r_2.x[j_p]-r_2.x[i_p]);
      dist += (r_2.y[j_p]-r_2.y[i_p])*(r_2.y[j_p]-r_2.y[i_p]);
      dist += (r_2.z[j_p]-r_2.z[i_p])*(r_2.z[j_p]-r_2.z[i_p]);
      dist = sqrt(dist);
      if (dist<(r_c*sig)){ accept = 0; break;}
    }
    float d_r = 0.0;
    d_r += r_2.x[i_p]*r_2.x[i_p];
    d_r += r_2.y[i_p]*r_2.y[i_p];
    d_r += r_2.z[i_p]*r_2.z[i_p];
    d_r = sqrt(d_r);
    if ((ap.R-d_r)<(r_c*sig)){ accept = 0;}

    //continue if it is accepted
    if (accept)
    {
      olddir.x = newdir.x;
      olddir.y = newdir.y;
      olddir.z = newdir.z;
      n_failures = 0;
    }
    else
    {
      ++n_failures;
      if( n_failures>1024){ i_p = 0;}
      else{ i_p--;}
    }
  }

  //free PRNG state
  hiprandDestroyGenerator(gen);
}

void chrsim::write_initial_configuration(FILE *f_ptr)
{
  std::fprintf(f_ptr,"Chromatin chrsim, t=0.0\n");
  std::fprintf(f_ptr,"%5d\n",ap.N);
  for( int i_p = 0; i_p<ap.N; ++i_p)
  {
    std::fprintf(f_ptr,"%5d%-5s%5s%5d",i_p+1,"X","X",i_p+1);
    std::fprintf(f_ptr,"%8.3f%8.3f%8.3f\n",r_2.x[i_p],r_2.y[i_p],r_2.z[i_p]);
  }
  std::fprintf(f_ptr,"%10.5f%10.5f%10.5f\n",0.0,0.0,0.0);
}

//Device Functions

//Global Functions

// __global__ void calc_bonded_f(int N, soa3 &r)
// {
//   int i_p = blockIdx.x * blockDim.x + threadIdx.x;
//   if (i_p<N)
//   {
//     r.x[i_p] = 0.0;
//   }
// }

} //namespace mmcc
