#include "hip/hip_runtime.h"
//Includes

#include "chrsim.cuh" //chromatin simulation
#include "util.hpp" //utilities

#include <time.h> //time utilities library

#include </usr/local/cuda/samples/common/inc/hip/hip_vector_types.h> //float4 utilities

//Namespace

namespace mmc //Marco Mendívil Carboni
{

//Constants

static constexpr float xi  = 1.000000; //damping coefficient
static constexpr float k_B = 0.001120; //Boltzmann constant
static constexpr float l_0 = 1.000000; //bond natural length
static constexpr float k_e = 100.0000; //elastic constant
static constexpr float k_b = 2.000000; //bending constant
static constexpr float r_c = 1.122462; //LJ cutoff radius
static constexpr float dt  = 1.0/2048; //timestep

//Device Functions

//calculate bonded forces
inline __device__ void calc_bonded_f(
  int N, //number of particles
  int i_p, //particle index
  float4 *r, //positions
  float4 *f) //forces
{
  //declare auxiliary variables
  float3 b_vec[4]; //bond vectors
  float b_i_l[4]; //bond inverse lengths
  float b_cos[3]; //bond angle cosines
  float3 f_b = make_float3(0.0); //bonded forces

  //calculate bond vectors, inverse lengths and angle cosines
  for (int i_b = 0; i_b<4; ++i_b) //bond index
  {
    if ((i_p+i_b)>=2 && (i_p+i_b)<=N) //calculate values if bond exists
    {
      b_vec[i_b] = make_float3(r[i_p+i_b-1]-r[i_p+i_b-2]);
      b_i_l[i_b] = rsqrtf(dot(b_vec[i_b],b_vec[i_b]));
    }
    else //set values to zero if bond doesn't exist
    {
      b_vec[i_b] = make_float3(0.0);
      b_i_l[i_b] = 0.0;
    }
  }
  for (int i_c = 0; i_c<3; ++i_c) //cosine index
  {
    b_cos[i_c] = dot(b_vec[i_c+1],b_vec[i_c])*b_i_l[i_c+1]*b_i_l[i_c];
  }

  //calculate elastic potential force
  f_b += k_e*(+(1.0-l_0*b_i_l[2])*b_vec[2]-(1.0-l_0*b_i_l[1])*b_vec[1]);

  //calculate bending potential force
  f_b += k_b*(+b_i_l[1]*b_i_l[0]*b_vec[0]-b_cos[0]*b_vec[1]*b_i_l[1]*b_i_l[1]);
  f_b += k_b*(+b_i_l[1]*b_i_l[2]*b_vec[2]-b_cos[1]*b_vec[1]*b_i_l[1]*b_i_l[1]);
  f_b += k_b*(-b_i_l[2]*b_i_l[1]*b_vec[1]+b_cos[1]*b_vec[2]*b_i_l[2]*b_i_l[2]);
  f_b += k_b*(-b_i_l[2]*b_i_l[3]*b_vec[3]+b_cos[2]*b_vec[2]*b_i_l[2]*b_i_l[2]);

  //add result to forces
  f[i_p] += make_float4(f_b);
}

//Global Functions

//initialize device PRNG state array
__global__ void init_PRNG(
  int N, //number of particles
  prng *state, //device PRNG state array
  int seed) //PRNG seed
{
  int i_p = blockIdx.x*blockDim.x+threadIdx.x; //particle index
  if (i_p<N)
  {
    hiprand_init(seed,i_p,0,&state[i_p]);
  }
}

//begin Runge-Kutta iteration
__global__ void begin_iter(
  int N, //number of particles
  float sd, //random number standard deviation
  float4 *f_2, //force array 2
  float4 *f_1, //force array 1
  float4 *n_r, //random number array
  prng *state) //device PRNG state array
{
  int i_p = blockIdx.x*blockDim.x+threadIdx.x; //particle index
  if (i_p<N)
  {
    n_r[i_p] = sd*hiprand_normal4(&state[i_p]);
    f_2[i_p] = make_float4(0.0);
    f_1[i_p] = make_float4(0.0);
  }
}

//execute 1st stage of the Runge-Kutta method
__global__ void exec_RK_1(
  int N, //number of particles
  float4 *r_2, //position array 2
  float4 *r_1, //position array 1
  float4 *f_2, //force array 2
  float4 *n_r) //random number array
{
  int i_p = blockIdx.x*blockDim.x+threadIdx.x; //particle index
  if (i_p<N)
  {
    calc_bonded_f(N,i_p,r_2,f_2);
    r_1[i_p] = r_2[i_p]+f_2[i_p]*dt/xi+n_r[i_p]/xi;
  }
}

//execute 2nd stage of the Runge-Kutta method
__global__ void exec_RK_2(
  int N, //number of particles
  float4 *r_2, //position array 2
  float4 *r_1, //position array 1
  float4 *f_2, //force array 2
  float4 *f_1, //force array 1
  float4 *n_r) //random number array
{
  int i_p = blockIdx.x*blockDim.x+threadIdx.x; //particle index
  if (i_p<N)
  {
    calc_bonded_f(N,i_p,r_1,f_1);
    r_2[i_p] = r_2[i_p]+0.5*(f_1[i_p]+f_2[i_p])*dt/xi+n_r[i_p]/xi;
  }
}

//Host Functions

//chrsim constructor
chrsim::chrsim(std::ifstream &f_par) //parameter file
{
  //initialize parameters and variables
  read_parameters(f_par);
  n_p_blk = (ap.N+thd_blk-1)/thd_blk;
  sd = sqrt(2.0*xi*k_B*ap.T*dt);

  //allocate unified memory
  cuda_check(hipMallocManaged(&r_2,ap.N*sizeof(float4)));
  cuda_check(hipMallocManaged(&r_1,ap.N*sizeof(float4)));
  cuda_check(hipMallocManaged(&f_2,ap.N*sizeof(float4)));
  cuda_check(hipMallocManaged(&f_1,ap.N*sizeof(float4)));
  cuda_check(hipMallocManaged(&n_r,ap.N*sizeof(float4)));
  cuda_check(hipMallocManaged(&state,ap.N*sizeof(prng)));

  //initialize PRNG
  init_PRNG<<<n_p_blk,thd_blk>>>(ap.N,state,time(nullptr));
  cuda_check(hipDeviceSynchronize());
}

//chrsim destructor
chrsim::~chrsim()
{
  hipFree(r_2);
  hipFree(r_1);
  hipFree(f_2);
  hipFree(f_1);
  hipFree(n_r);
  hipFree(state);
}

//generate a random initial condition
void chrsim::generate_initial_condition()
{
  //initialize PRNG
  hiprandGenerator_t gen; //host PRNG
  hiprandCreateGeneratorHost(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen,time(nullptr));

  //declare auxiliary variables
  float beta = 1.0/(k_B*ap.T); //inverse temperature
  float rand; //random number in (0,1]
  float theta; //polar angle
  float phi; //azimuthal angle
  float len_b; //bond length
  float angle_b; //bond angle
  float3 olddir; //old direction
  float3 newdir; //new direction
  float3 randdir; //random direction
  float3 perpdir; //perpendicular direction

  //place first particle
  r_2[0] = make_float4(0.0);
  hiprandGenerateUniform(gen,&rand,1); theta = acos(1.0-2.0*rand);
  hiprandGenerateUniform(gen,&rand,1); phi = 2.0*M_PI*rand;
  randdir = make_float3(sin(theta)*cos(phi),sin(theta)*sin(phi),cos(theta));
  olddir = randdir;

  //reduce sigma for random walk
  sig = 1.0/2;

  //perform random walk
  int att = 0; //number of attempts
  for (int i_p = 1; i_p<ap.N; ++i_p) //particle index
  {
    //generate random direction perpendicular to old direction
    hiprandGenerateUniform(gen,&rand,1); theta = acos(1.0-2.0*rand);
    hiprandGenerateUniform(gen,&rand,1); phi = 2.0*M_PI*rand;
    randdir = make_float3(sin(theta)*cos(phi),sin(theta)*sin(phi),cos(theta));
    perpdir = cross(olddir,randdir);
    perpdir = normalize(perpdir);

    //generate random bond angle and calculate new direction
    hiprandGenerateUniform(gen,&rand,1);
    angle_b = acos(1.0+log(1.0-(1.0-exp(-2.0*(k_b*beta)))*rand)/(k_b*beta));
    newdir = cos(angle_b)*olddir+sin(angle_b)*perpdir;

    //calculate position of next particle
    hiprandGenerateUniform(gen,&rand,1);
    len_b = l_0+sqrt(2.0/(k_e*beta))*erfinv(2.0*rand-1.0);
    r_2[i_p] = make_float4(len_b*newdir)+r_2[i_p-1];

    //check if new position is acceptable
    bool p_a = true; //position is acceptable
    if (!isfinite(r_2[i_p].x)){ p_a = false;}
    if (!isfinite(r_2[i_p].y)){ p_a = false;}
    if (!isfinite(r_2[i_p].z)){ p_a = false;}
    for (int j_p = 0; j_p<(i_p-1); ++j_p) //secondary particle index
    {
      float d_pp; //particle-particle distance
      d_pp = length(make_float3(r_2[j_p]-r_2[i_p]));
      if (d_pp<(r_c*sig)){ p_a = false; break;}
    }
    float d_r; //radial distance to origin
    d_r = length(make_float3(r_2[i_p]));
    if ((ap.R-d_r)<(r_c*sig)){ p_a = false;}

    if (p_a) //continue
    {
      olddir = newdir;
      att = 0;
    }
    else //go back
    {
      ++att;
      if (att>1024){ i_p = 1;}
      else{ i_p--;}
    }
  }

  //expand beads and reset sigma
  while (sig<1.0)
  {
    make_RK_iteration();
    sig *= 1.0+dt/4;
  }
  cuda_check(hipDeviceSynchronize());
  sig = 1.0;

  //free host PRNG
  hiprandDestroyGenerator(gen);

  //record success message
  logger::record("initial condition generated");
}

//write initial condition to file in gro format
void chrsim::write_initial_condition(std::ofstream &f_ic) //IC file
{
  f_ic<<"Chromatin simulation, i_f = 0, t = 0.0\n";
  f_ic<<cnfs(ap.N,5,' ')<<"\n";
  for (int i_p = 0; i_p<ap.N; ++i_p) //particle index
  {
    f_ic<<std::setw(5)<<i_p+1<<std::left<<std::setw(5)<<"X"<<std::right;
    f_ic<<std::setw(5)<<"X"<<std::setw(5)<<i_p+1;
    f_ic<<cnfs(r_2[i_p].x,8,' ',3);
    f_ic<<cnfs(r_2[i_p].y,8,' ',3);
    f_ic<<cnfs(r_2[i_p].z,8,' ',3);
    f_ic<<"\n";
  }
  f_ic<<cnfs(0.0,10,' ',5);
  f_ic<<cnfs(0.0,10,' ',5);
  f_ic<<cnfs(0.0,10,' ',5);
  f_ic<<"\n";
}

//save simulation state to binary file
void chrsim::save_checkpoint(std::ofstream &f_chkp) //checkpoint file
{
  f_chkp.write(reinterpret_cast<char *>(&i_f),sizeof(i_f));
  f_chkp.write(reinterpret_cast<char *>(&t),sizeof(t));
  f_chkp.write(reinterpret_cast<char *>(r_2),ap.N*sizeof(float4));
  f_chkp.write(reinterpret_cast<char *>(state),ap.N*sizeof(prng));
  logger::record("simulation checkpoint saved");
}

//load simulation state from binary file
void chrsim::load_checkpoint(std::ifstream &f_chkp) //checkpoint file
{
  f_chkp.read(reinterpret_cast<char *>(&i_f),sizeof(i_f));
  f_chkp.read(reinterpret_cast<char *>(&t),sizeof(t));
  f_chkp.read(reinterpret_cast<char *>(r_2),ap.N*sizeof(float4));
  f_chkp.read(reinterpret_cast<char *>(state),ap.N*sizeof(prng));
  logger::record("simulation checkpoint loaded");
}

//run simulation and write trajectory file
void chrsim::run_simulation(std::ofstream &f_traj) //trajectory file
{
  for (int f = 0; f<ap.f_f; ++f) //frame
  {
    float prog_pc = (100.0*f)/(ap.f_f); //progress percentage
    logger::show_prog_pc(prog_pc);
    for (int s = 0; s<ap.f_s; ++s) //step
    {
      make_RK_iteration();
    }
    cuda_check(hipDeviceSynchronize());
    ++i_f; t += ap.f_s*dt;
    write_trajectory_frame(f_traj);
  }
}

//read adjustable parameters from file
void chrsim::read_parameters(std::ifstream &f_par) //parameter file
{
  std::string key; //parameter string key
  f_par>>key>>(ap.T); if (key!="T"||ap.T<0){ throw error("error reading T");}
  f_par>>key>>(ap.N); if (key!="N"||ap.N<1){ throw error("error reading N");}
  f_par>>key>>(ap.R); if (key!="R"||ap.R<0){ throw error("error reading R");}
  f_par>>key>>(ap.f_f); if (key!="F"||ap.f_f<1){ throw error("error reading F");}
  std::string msg = "parameters:"; //message
  msg += " T = "+cnfs(ap.T,6,'0',2);
  msg += " N = "+cnfs(ap.N,5,'0');
  msg += " R = "+cnfs(ap.R,6,'0',2);
  msg += " F = "+cnfs(ap.f_f,5,'0');
  logger::record(msg);
  float cvf = ap.N*pow(0.5*sig/(ap.R-0.5*sig),3); //chromatin volume fraction
  if (cvf>0.5){ throw error("chromatin volume fraction above 0.5");}
}

//make one iteration of the Runge-Kutta method
void chrsim::make_RK_iteration()
{
  begin_iter<<<n_p_blk,thd_blk>>>(ap.N,sd,f_2,f_1,n_r,state);
  exec_RK_1<<<n_p_blk,thd_blk>>>(ap.N,r_2,r_1,f_2,n_r);
  exec_RK_2<<<n_p_blk,thd_blk>>>(ap.N,r_2,r_1,f_2,f_1,n_r);
}

//write trajectory frame to binary file in trr format
void chrsim::write_trajectory_frame(std::ofstream &f_traj) //trajectory file
{
  //this is a minimal trr file writing routine that doesn't rely on \ 
  //the xdr library but only works with vmd in little endian systems

  uint32_t header[18] = {1993, 1, 0, 
    0, 0, 0, 0, 0, 0, 0, 3*ap.N*4, 0, 0, ap.N, i_f, 0, 
    *(reinterpret_cast<uint32_t *>(&t)), 0}; //frame header
  //for more information on the contents of the header see chemfiles
  f_traj.write(reinterpret_cast<char *>(header),sizeof(header));
  for (int i_p = 0; i_p<ap.N; ++i_p) //particle index
  {
    f_traj.write(reinterpret_cast<char *>(&(r_2[i_p].x)),4);
    f_traj.write(reinterpret_cast<char *>(&(r_2[i_p].y)),4);
    f_traj.write(reinterpret_cast<char *>(&(r_2[i_p].z)),4);
  }
}

//check for errors in cuda runtime API call
void cuda_check(hipError_t rtn_val) //cuda runtime API call return value
{
  if (rtn_val!=hipSuccess)
  {
    std::string msg = "cuda: "; //error message
    msg += hipGetErrorString(rtn_val);
    throw error(msg);
  }
}

} //namespace mmc
