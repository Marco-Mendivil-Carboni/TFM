#include "hip/hip_runtime.h"
//Includes

#include "chrsim.cuh" //chromatin simulation
#include "util.hpp" //utilities

#include <time.h> //time utilities library
#include </usr/local/cuda/samples/common/inc/hip/hip_vector_types.h> //float4 utilities

//Namespace

namespace mmcc //Marco Mendívil Carboni code
{

//Constants

static constexpr float xi  = 1.000000; //damping coefficient
static constexpr float k_B = 0.001120; //Boltzmann constant
static constexpr float l_0 = 1.000000; //bond natural length
static constexpr float k_e = 100.0000; //elastic constant
static constexpr float k_b = 2.000000; //bending constant
static constexpr float r_c = 1.122462; //LJ cutoff radius
static constexpr float dt  = 1.0/2048; //timestep

static constexpr int f_s = 1*2048; //RK steps per frame

//Device Functions

//Global Functions

//initialize device PRNG state
__global__ void init_PRNG(
  prng *state, //device PRNG state
  int seed) //PRNG seed
{
  int i_p = blockIdx.x*blockDim.x+threadIdx.x; //particle index
  hiprand_init(seed,i_p,0,&state[i_p]);
}

//begin Runge-Kutta iteration
__global__ void begin_iter(
  int N, //number of particles
  float c_rn, //random number constant
  float4 *f_2, //forces 2
  float4 *f_1, //forces 1
  float4 *nrn, //normal random numbers
  prng *state) //device PRNG state
{
  int i_p = blockIdx.x*blockDim.x+threadIdx.x; //particle index
  float4 nrn_thd = nrn[i_p]; //thread normal random numbers
  nrn_thd.x = c_rn*hiprand_normal(&state[i_p]);
  nrn_thd.y = c_rn*hiprand_normal(&state[i_p]);
  nrn_thd.z = c_rn*hiprand_normal(&state[i_p]);
  nrn[i_p] = nrn_thd;
  if (i_p<N)
  {
    f_2[i_p] = make_float4(0.0);
    f_1[i_p] = make_float4(0.0);
  }
}

//execute 1st stage of the Runge-Kutta method
__global__ void exec_RK_1(
  int N, //number of particles
  float4 *r_2, //positions 2
  float4 *r_1, //positions 1
  float4 *f_2, //forces 2
  float4 *nrn) //normal random numbers
{
  int i_p = blockIdx.x*blockDim.x+threadIdx.x; //particle index
  if (i_p<N)
  {

  }
}

//execute 2nd stage of the Runge-Kutta method
__global__ void exec_RK_2(
  int N, //number of particles
  float4 *r_2, //positions 2
  float4 *r_1, //positions 1
  float4 *f_2, //forces 2
  float4 *f_1, //forces 1
  float4 *nrn) //normal random numbers
{
  int i_p = blockIdx.x*blockDim.x+threadIdx.x; //particle index
  if (i_p<N)
  {

  }
}

//Host Functions

//check for errors in cuda runtime API call
void cuda_check(hipError_t rtn_val) //cuda runtime API call return value
{
  if (rtn_val!=hipSuccess)
  {
    std::string msg = "cuda: "; //error message
    msg += hipGetErrorString(rtn_val);
    throw error(msg);
  }
}

//chrsim constructor
chrsim::chrsim(std::ifstream &f_par) //parameter file
{
  //initialize parameters and variables
  read_parameters(f_par);
  n_p_blk = (ap.N+thd_blk-1)/thd_blk;
  n_p_thd = n_p_blk*thd_blk;
  c_rn = sqrt(2.0*xi*k_B*ap.T*dt);

  //allocate unified memory
  cuda_check(hipMallocManaged(&r_2,ap.N*sizeof(float4)));
  cuda_check(hipMallocManaged(&r_1,ap.N*sizeof(float4)));
  cuda_check(hipMallocManaged(&f_2,ap.N*sizeof(float4)));
  cuda_check(hipMallocManaged(&f_1,ap.N*sizeof(float4)));
  cuda_check(hipMallocManaged(&nrn,n_p_thd*sizeof(float4)));
  cuda_check(hipMallocManaged(&state,n_p_thd*sizeof(prng)));

  //initialize PRNG
  init_PRNG<<<n_p_blk,thd_blk>>>(state,time(nullptr));
  cuda_check(hipDeviceSynchronize());
}

//chrsim destructor
chrsim::~chrsim()
{
  hipFree(r_2);
  hipFree(r_1);
  hipFree(f_2);
  hipFree(f_1);
  hipFree(nrn);
  hipFree(state);
}

//generate a random initial condition
void chrsim::generate_initial_condition()
{
  //initialize PRNG
  hiprandGenerator_t gen; //host PRNG
  hiprandCreateGeneratorHost(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen,time(nullptr));

  //declare auxiliary variables
  float beta = 1.0/(k_B*ap.T); //inverse temperature
  float rand; //random number in (0,1]
  float theta; //polar angle
  float phi; //azimuthal angle
  float len_b; //bond length
  float angle_b; //bond angle
  float3 randdir; //random direction
  float3 olddir; //old direction
  float3 newdir; //new direction
  float3 perpdir; //perpendicular direction

  //place first particle
  r_2[0] = make_float4(0.0);
  hiprandGenerateUniform(gen,&rand,1); theta = acos(1.0-2.0*rand);
  hiprandGenerateUniform(gen,&rand,1); phi = 2.0*M_PI*rand;
  randdir = make_float3(sin(theta)*cos(phi),sin(theta)*sin(phi),cos(theta));
  olddir = randdir;

  //set initial sigma value
  sig = 1.0/2;

  //perform random walk
  int att = 0; //number of attempts
  for (int i_p = 1; i_p<ap.N; ++i_p) //particle index
  {
    //generate random direction perpendicular to old direction
    hiprandGenerateUniform(gen,&rand,1); theta = acos(1.0-2.0*rand);
    hiprandGenerateUniform(gen,&rand,1); phi = 2.0*M_PI*rand;
    randdir = make_float3(sin(theta)*cos(phi),sin(theta)*sin(phi),cos(theta));
    perpdir = cross(olddir,randdir);
    perpdir = normalize(perpdir);

    //generate random bond angle and calculate new direction
    hiprandGenerateUniform(gen,&rand,1);
    angle_b = acos(1.0+log(1.0-(1.0-exp(-2.0*(k_b*beta)))*rand)/(k_b*beta));
    newdir = cos(angle_b)*olddir+sin(angle_b)*perpdir;

    //calculate position of next particle
    hiprandGenerateUniform(gen,&rand,1);
    len_b = l_0+sqrt(2.0/(k_e*beta))*erfinv(2.0*rand-1.0);
    r_2[i_p] = make_float4(len_b*newdir)+r_2[i_p-1];

    //check if new position is acceptable
    bool p_a = true; //position is acceptable
    if (!isfinite(r_2[i_p].x)){ p_a = false;}
    if (!isfinite(r_2[i_p].y)){ p_a = false;}
    if (!isfinite(r_2[i_p].z)){ p_a = false;}
    for (int j_p = 0; j_p<(i_p-1); ++j_p) //secondary particle index
    {
      float d_pp; //particle-particle distance
      d_pp = length(make_float3(r_2[j_p]-r_2[i_p]));
      if (d_pp<(r_c*sig)){ p_a = false; break;}
    }
    float d_r; //radial distance to origin
    d_r = length(make_float3(r_2[i_p]));
    if ((ap.R-d_r)<(r_c*sig)){ p_a = false;}

    if (p_a) //continue
    {
      olddir = newdir;
      att = 0;
    }
    else //go back
    {
      ++att;
      if (att>1024){ i_p = 1;}
      else{ i_p--;}
    }
  }

  //free host PRNG
  hiprandDestroyGenerator(gen);

  //record success message
  logger::record("initial condition generated");
}

//write initial condition to file in gro format
void chrsim::write_initial_condition(std::ofstream &f_i_c) //initial condition file
{
  f_i_c<<"Chromatin simulation, i_f = 0, t = 0.0\n";
  f_i_c<<cnfs(ap.N,5,' ')<<"\n";
  for (int i_p = 0; i_p<ap.N; ++i_p) //particle index
  {
    f_i_c<<std::setw(5)<<i_p+1<<std::left<<std::setw(5)<<"X"<<std::right;
    f_i_c<<std::setw(5)<<"X"<<std::setw(5)<<i_p+1;
    f_i_c<<cnfs(r_2[i_p].x,8,' ',3);
    f_i_c<<cnfs(r_2[i_p].y,8,' ',3);
    f_i_c<<cnfs(r_2[i_p].z,8,' ',3);
    f_i_c<<"\n";
  }
  f_i_c<<cnfs(0.0,10,' ',5);
  f_i_c<<cnfs(0.0,10,' ',5);
  f_i_c<<cnfs(0.0,10,' ',5);
  f_i_c<<"\n";
}

//save simulation state to binary file
void chrsim::save_checkpoint(std::ofstream &f_chkp) //checkpoint file
{
  f_chkp.write(reinterpret_cast<char *>(&i_f),sizeof(i_f));
  f_chkp.write(reinterpret_cast<char *>(&t),sizeof(t));
  f_chkp.write(reinterpret_cast<char *>(r_2),ap.N*sizeof(float4));
  f_chkp.write(reinterpret_cast<char *>(state),n_p_thd*sizeof(prng));
  logger::record("simulation checkpoint saved");
}

//load simulation state from binary file
void chrsim::load_checkpoint(std::ifstream &f_chkp) //checkpoint file
{
  f_chkp.read(reinterpret_cast<char *>(&i_f),sizeof(i_f));
  f_chkp.read(reinterpret_cast<char *>(&t),sizeof(t));
  f_chkp.read(reinterpret_cast<char *>(r_2),ap.N*sizeof(float4));
  f_chkp.read(reinterpret_cast<char *>(state),n_p_thd*sizeof(prng));
  logger::record("simulation checkpoint loaded");
}

//run simulation and write trajectory file
void chrsim::run_simulation(std::ofstream &f_traj) //trajectory file
{
  for (int f = 0; f<ap.F; ++f) //frame
  {
    float prog_pc = (100.0*f)/(ap.F); //progress percentage
    mmcc::logger::show_prog_pc(prog_pc);
    for (int s = 0; s<f_s; ++s) //step
    {
      take_step();
    }
    cuda_check(hipDeviceSynchronize());
    ++i_f;
    t += f_s*dt;
    write_trajectory_frame(f_traj);
  }
}

//read adjustable parameters from file
void chrsim::read_parameters(std::ifstream &f_par) //parameter file
{
  std::string key; //parameter string key
  f_par>>key>>(ap.T); if (key!="T"||ap.T<0){ throw error("error reading T");}
  f_par>>key>>(ap.N); if (key!="N"||ap.N<1){ throw error("error reading N");}
  f_par>>key>>(ap.R); if (key!="R"||ap.R<0){ throw error("error reading R");}
  f_par>>key>>(ap.F); if (key!="F"||ap.F<1){ throw error("error reading F");}
  std::string msg = "parameters:"; //message
  msg += " T = "+cnfs(ap.T,6,'0',2);
  msg += " N = "+cnfs(ap.N,5,'0');
  msg += " R = "+cnfs(ap.R,6,'0',2);
  msg += " F = "+cnfs(ap.F,5,'0');
  logger::record(msg);
  float cvf = ap.N*pow(0.5/(ap.R-0.5),3); //chromatin volume fraction
  if (cvf>0.5){ throw error("chromatin volume fraction above 0.5");}
}

//take RK step------------------------------------------------------------------tmp
void chrsim::take_step()//------------------------------------------------------tmp
{
  begin_iter<<<n_p_blk,thd_blk>>>(ap.N,c_rn,f_2,f_1,nrn,state);
}

//write trajectory frame to binary file in trr format
void chrsim::write_trajectory_frame(std::ofstream &f_traj) //trajectory file
{
  int32_t header[] = {1993, 13, 12, 
    1599622471, 1601073780, 1701603686, 
    0, 0, 0, 0, 0, 0, 0, 3*ap.N*4, 0, 0, ap.N, i_f, 0, 
    *(reinterpret_cast<int32_t *>(&t)), 0}; //trr file header
  //for more information on the contents of the header see chemfiles
  f_traj.write(reinterpret_cast<char *>(header),sizeof(header));
  for (int i_p = 0; i_p<ap.N; ++i_p) //particle index
  {
    f_traj.write(reinterpret_cast<char *>(&(r_2[i_p].x)),4);
    f_traj.write(reinterpret_cast<char *>(&(r_2[i_p].y)),4);
    f_traj.write(reinterpret_cast<char *>(&(r_2[i_p].z)),4);
  }
  //this is a minimal trr file writing routine that doesn't rely on \ 
  //the xdr library but only works with vmd in little endian systems
}

// __device__ void example_function(float3 &r)
// {
//   r += ...;
// }
// __global__ void example_kernel(int N, float4 *r)
// {
//   int i_p = ...;
//   if (i_p<N)
//   {
//     float3 r_int = make_float3(r[i_p]);
//     use r_int by reference in device functions onward
//     example_function(r_int);
//   }
// }

} //namespace mmcc
