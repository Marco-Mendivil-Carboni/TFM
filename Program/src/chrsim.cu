#include "hip/hip_runtime.h"
//Includes

#include "chrsim.cuh" //chromatin simulation

#include <time.h> //time utilities library

#include <hiprand/hiprand_kernel.h> //cuRAND device functions

//Namespace

namespace mmc //Marco Mendívil Carboni
{

//Constants

static constexpr float dt = 1.0/2048; //timestep
static constexpr float rco = 1.154701; //repulsive cutoff
static constexpr float aco = 2.000000; //attractive cutoff
static constexpr float mis = 0.838732; //minimum initial separation

//Aliases

using prng = hiprandStatePhilox4_32_10; //PRNG type

//Enumerations

enum stype //simulation type
{
  DST, //default simulation type
  ICG //initial condition generation
};

//Device Functions

//calculate bonded forces
inline __device__ void calc_bf(
  const uint N, //number of particles
  uint i_p, //particle index
  float4 *r, //position array
  float4 *f) //force array
{
  //declare auxiliary variables
  float3 vec[4]; //bond vectors
  float il[4]; //bond inverse lengths
  float cos[3]; //bond angle cosines
  float3 bf = {0.0,0.0,0.0}; //bonded forces

  //calculate bond vectors, inverse lengths and angle cosines
  for (uint i_b = 0; i_b<4; ++i_b) //bond index
  {
    if ((i_p+i_b)>=2 && (i_p+i_b)<=N) //calculate variables if bond exists
    {
      vec[i_b] = make_float3(r[i_p+i_b-1]-r[i_p+i_b-2]);
      il[i_b] = rsqrtf(dot(vec[i_b],vec[i_b]));
    }
    else //set variables to zero if bond doesn't exist
    {
      vec[i_b] = {0.0,0.0,0.0};
      il[i_b] = 0.0;
    }
  }
  for (uint i_c = 0; i_c<3; ++i_c) //cosine index
  {
    cos[i_c] = dot(vec[i_c+1],vec[i_c])*il[i_c+1]*il[i_c];
  }

  //calculate elastic potential force
  bf += k_e*(+(1.0-l_0*il[2])*vec[2]-(1.0-l_0*il[1])*vec[1]);

  //calculate bending potential force
  bf += k_b*(+il[1]*il[0]*vec[0]-cos[0]*vec[1]*il[1]*il[1]);
  bf += k_b*(+il[1]*il[2]*vec[2]-cos[1]*vec[1]*il[1]*il[1]);
  bf += k_b*(-il[2]*il[1]*vec[1]+cos[1]*vec[2]*il[2]*il[2]);
  bf += k_b*(-il[2]*il[3]*vec[3]+cos[2]*vec[2]*il[2]*il[2]);

  //add result to force array
  f[i_p] += make_float4(bf);
}

//calculate confinement force
inline __device__ void calc_cf(
  const float R, //confinement radius
  uint i_p, //particle index
  float4 *r, //position array
  float4 *f) //force array
{
  //calculate auxiliary variables
  float d_r; //radial distance to origin
  d_r = length(make_float3(r[i_p]));
  float dwp = R-d_r; //wall-particle distance
  if (dwp>rco){ return;}

  //calculate confinement force
  float3 cf = make_float3(-r[i_p]/d_r); //confinement force
  float d2 = dwp*dwp; //dwp squared
  cf *= (18.0*d2*d2-96.0*d2+96.0)/(d2*d2*d2*d2);

  //add result to force array
  f[i_p] += make_float4(cf);
}

//calculate particle force
template <stype T> inline __device__ void calc_pf(
  const float eps, //particle energy
  float3 vpp, //particle particle vector
  float dpp, //particle particle distance
  float3 &srf) //short-range forces
{
  //calculate Wang-Frenkel force
  if (dpp>aco){ return;}
  float d2 = dpp*dpp; //dpp squared
  srf += eps*(18.0*d2*d2-96.0*d2+96.0)/(d2*d2*d2*d2)*vpp;
}

//calculate particle force
template <> inline __device__ void calc_pf<ICG>(
  const float eps, //particle energy
  float3 vpp, //particle particle vector
  float dpp, //particle particle distance
  float3 &srf) //short-range forces
{
  //calculate Soft-Repulsive force
  if (dpp>rco){ return;}
  srf += 128.0*(3.0*rco-3.0*dpp)*vpp;
}

//calculate lbs force
template <stype T> inline __device__ void calc_lf(
  float3 vlp, //lbs particle vector
  float dlp, //lbs particle distance
  float3 &srf) //short-range forces
{
  //calculate lbs force
  if (dlp>0.5){ return;}
  float d6 = dlp*dlp*dlp*dlp*dlp*dlp; //dlp to the sixth power
  float r6 = 0.5*0.5*0.5*0.5*0.5*0.5; //r_c to the sixth power
  srf += 128.0*(d6/r6-1.0)*vlp;
}

//calculate lbs force
template <> inline __device__ void calc_lf<ICG>(
  float3 vpp, //lbs particle vector
  float dpp, //lbs particle distance
  float3 &srf) //short-range forces
{
  return;
}

//calculate short-range forces with cell's objects
template <stype T> inline __device__ void calc_cell_srf(
  const float eps, //particle energy
  ptype *pt, //particle type array
  float4 *lr, //lbs position array
  uint i_c, //cell index
  uint i_p, //particle index
  float3 r_i, //particle position
  float4 *r, //position array
  sugrid *pgp, //particle grid pointer
  sugrid *lgp, //lbs grid pointer
  float3 &srf) //short-range forces
{
  //declare auxiliary variables
  uint j_p; //secondary particle index
  uint i_l; //lbs index
  uint beg; //cell beginning
  uint end; //cell end

  //get particle grid limits
  beg = pgp->beg[i_c]; //cell beginning
  end = pgp->end[i_c]; //cell end

  //range over cell's particles
  for (uint sai = beg; sai<end; ++sai) //sorted array index
  {
    //get secondary particle index
    j_p = pgp->spi[sai];

    //calculate force only between non-bonded particles
    if (((j_p>i_p)?j_p-i_p:i_p-j_p)>1)
    {
      //calculate particle particle distance
      float3 vpp = r_i-make_float3(r[j_p]); //particle particle vector
      float dpp = length(vpp); //particle particle distance

      //calculate particle force
      calc_pf<T>(eps,vpp,dpp,srf);
    }
  }

  //calculate lbs force only for LAD particles
  if (pt[i_p]==LAD)
  {
    //get lbs grid limits
    beg = lgp->beg[i_c];
    end = lgp->end[i_c];

    //range over cell's lbs
    for (uint sai = beg; sai<end; ++sai) //sorted array index
    {
      //get lbs index
      i_l = lgp->spi[sai];

      //calculate lbs particle distance
      float3 vlp = r_i-make_float3(lr[i_l]); //lbs particle vector
      float dlp = length(vlp); //lbs particle distance

      //calculate lbs force
      calc_lf<T>(vlp,dlp,srf);
    }
  }
}

//calculate all short-range forces
template <stype T> inline __device__ void calc_all_srf(
  const float eps, //particle energy
  ptype *pt, //particle type array
  float4 *lr, //lbs position array
  uint i_p, //particle index
  float4 *r, //position array
  float4 *f, //force array
  sugrid *pgp, //particle grid pointer
  sugrid *lgp) //lbs grid pointer
{
  //calculate auxiliary variables
  float3 r_i = make_float3(r[i_p]); //particle position
  const float csl = pgp->csl; //cell side length
  const uint cps = pgp->cps; //cells per side
  const uint n_c = pgp->n_c; //number of cells
  int3 ir = floorf(r_i/csl); //integer coordinates
  uint iofst = (cps/2)*(1+cps+cps*cps); //index offset
  float3 srf = {0.0,0.0,0.0}; //short-range forces

  //range over neighbouring cells
  uint nci; //neighbour cell index
  int3 nir; //neighbour integer coordinates
  for (nir.x = ir.x-1; nir.x<=ir.x+1; ++nir.x)
  {
    for (nir.y = ir.y-1; nir.y<=ir.y+1; ++nir.y)
    {
      for (nir.z = ir.z-1; nir.z<=ir.z+1; ++nir.z)
      {
        //calculate neighbour cell index
        nci = iofst+nir.x+nir.y*cps+nir.z*cps*cps;
        if (nci>=n_c){ continue;}

        //calculate short-range forces with cell's objects
        calc_cell_srf<T>(eps,pt,lr,nci,i_p,r_i,r,pgp,lgp,srf);
      }
    }
  }

  //add result to force array
  f[i_p] += make_float4(srf);
}

//Global Functions

//initialize PRNG state array
__global__ void init_ps(
  const uint N, //number of particles
  void *vps, //void PRNG state array
  uint pseed) //PRNG seed
{
  //calculate particle index
  uint i_p = blockIdx.x*blockDim.x+threadIdx.x; //particle index
  if (i_p>=N){ return;}

  //initialize PRNG state
  prng *ps = static_cast<prng *>(vps); //PRNG state array
  hiprand_init(pseed,i_p,0,&ps[i_p]);
}

//execute 1st stage of the Runge-Kutta method
template <stype T> __global__ void exec_RK_1(
  const uint N, //number of particles
  const float R, //confinement radius
  const float eps, //particle energy
  ptype *pt, //particle type array
  float4 *r, //position array
  float4 *f, //force array
  float4 *lr, //lbs position array
  float4 *er, //extra position array
  float sd, //standard deviation
  float4 *rn, //random number array
  void *vps, //void PRNG state array
  sugrid *pgp, //particle grid pointer
  sugrid *lgp) //lbs grid pointer
{
  //calculate particle index
  uint i_p = blockIdx.x*blockDim.x+threadIdx.x; //particle index
  if (i_p>=N){ return;}

  //calculate random numbers
  prng *ps = static_cast<prng *>(vps); //PRNG state array
  float3 az; //absolute z-score
  do
  {
    rn[i_p] = sd*hiprand_normal4(&ps[i_p]);
    az = fabs(make_float3(rn[i_p])/sd);
  }
  while (az.x>5||az.y>5||az.z>5);

  //calculate forces
  f[i_p] = {0.0,0.0,0.0,0.0};
  calc_bf(N,i_p,r,f);
  calc_cf(R,i_p,r,f);
  calc_all_srf<T>(eps,pt,lr,i_p,r,f,pgp,lgp);

  //calculate extra position
  er[i_p] = r[i_p]+f[i_p]*dt+rn[i_p];
}

//execute 2nd stage of the Runge-Kutta method
template <stype T> __global__ void exec_RK_2(
  const uint N, //number of particles
  const float R, //confinement radius
  const float eps, //particle energy
  ptype *pt, //particle type array
  float4 *r, //position array
  float4 *f, //force array
  float4 *lr, //lbs position array
  float4 *er, //extra position array
  float4 *ef, //extra force array
  float4 *rn, //random number array
  sugrid *pgp, //particle grid pointer
  sugrid *lgp) //lbs grid pointer
{
  //calculate particle index
  uint i_p = blockIdx.x*blockDim.x+threadIdx.x; //particle index
  if (i_p>=N){ return;}

  //calculate forces
  ef[i_p] = {0.0,0.0,0.0,0.0};
  calc_bf(N,i_p,er,ef);
  calc_cf(R,i_p,er,ef);
  calc_all_srf<T>(eps,pt,lr,i_p,er,ef,pgp,lgp);

  //calculate new position
  r[i_p] = r[i_p]+0.5*(ef[i_p]+f[i_p])*dt+rn[i_p];
}

//Host Functions

//chromatin simulation constructor
chrsim::chrsim(parmap &par) //parameters
  : chrdat(par)
  , fpf {par.get_val<uint>("frames_per_file",100)}
  , spf {par.get_val<uint>("steps_per_frame",1*2048)}
  , tpb {par.get_val<uint>("threads_per_block",256)}
  , sd {static_cast<float>(sqrt(2.0*k_B*T*dt))}
  , pg(N,aco,2*ceil(R/aco))
  , lg(n_l,pg)
{
  //check parameters
  if (!(1<=fpf&&fpf<10'000)){ throw error("frames_per_file out of range");}
  if (!(1<=spf&&spf<10'000)){ throw error("steps_per_frame out of range");}
  if (!(1<=tpb&&tpb<1'025)){ throw error("threads_per_block out of range");}
  std::string msg = ""; //message
  msg += "fpf = "+cnfs(fpf,4,'0')+" ";
  msg += "spf = "+cnfs(spf,4,'0')+" ";
  msg += "tpb = "+cnfs(tpb,4,'0')+" ";
  logger::record(msg);

  //allocate device memory
  cuda_check(hipMalloc(&er,N*sizeof(float4)));
  cuda_check(hipMalloc(&ef,N*sizeof(float4)));
  cuda_check(hipMalloc(&rn,N*sizeof(float4)));
  cuda_check(hipMalloc(&vps,N*sizeof(prng)));
  cuda_check(hipMalloc(&pgp,sizeof(sugrid)));
  cuda_check(hipMalloc(&lgp,sizeof(sugrid)));

  //copy grids to device
  cuda_check(hipMemcpy(pgp,&pg,sizeof(sugrid),hipMemcpyHostToDevice));
  cuda_check(hipMemcpy(lgp,&lg,sizeof(sugrid),hipMemcpyHostToDevice));

  //initialize PRNG
  init_ps<<<(N+tpb-1)/tpb,tpb>>>(N,vps,time(nullptr));
}

//chromatin simulation destructor
chrsim::~chrsim()
{
  //deallocate device memory
  cuda_check(hipFree(er));
  cuda_check(hipFree(ef));
  cuda_check(hipFree(rn));
  cuda_check(hipFree(vps));
  cuda_check(hipFree(pgp));
  cuda_check(hipFree(lgp));
}

//generate a random initial condition
void chrsim::generate_initial_condition()
{
  //set random lbs positions
  set_lbs_positions();

  //set random particle types
  set_particle_types();

  //perform a confined random walk
  perform_random_walk();

  //separate beads
  uint po = particle_overlaps(); //particle overlaps
  uint ipo = po; //initial particle overlaps
  while (po>0)
  {
    //show separation progress
    logger::show_prog_pc(100.0*(ipo-po)/ipo);

    //iterate over all steps per frame
    for (uint fsi = 0; fsi<spf; ++fsi) //frame step index
    {
      //make one Runge-Kutta iteration
      pg.generate_arrays(tpb,r);
      exec_RK_1<ICG><<<(N+tpb-1)/tpb,tpb>>>
      (N,R,eps,pt,r,f,lr,er,sd,rn,vps,pgp,lgp);
      pg.generate_arrays(tpb,er);
      exec_RK_2<ICG><<<(N+tpb-1)/tpb,tpb>>>
      (N,R,eps,pt,r,f,lr,er,ef,rn,pgp,lgp);
    }

    //copy position array to host
    cuda_check(hipMemcpy(hr,r,N*sizeof(float4),hipMemcpyDeviceToHost));

    //count particle overlaps
    po = particle_overlaps();
  }

  //record success message
  logger::record("initial condition generated");
}

//save simulation state to binary file
void chrsim::save_checkpoint(std::ofstream &bin_out_f) //binary output file
{
  //write simulation data
  bin_out_f.write(reinterpret_cast<char *>(&i_f),sizeof(i_f));
  bin_out_f.write(reinterpret_cast<char *>(&t),sizeof(t));
  bin_out_f.write(reinterpret_cast<char *>(hpt),N*sizeof(ptype));
  bin_out_f.write(reinterpret_cast<char *>(hr),N*sizeof(float4));
  bin_out_f.write(reinterpret_cast<char *>(hlr),n_l*sizeof(float4));

  //record success message
  logger::record("simulation checkpoint saved");
}

//load simulation state from binary file
void chrsim::load_checkpoint(std::ifstream &bin_inp_f) //binary input file
{
  //read simulation data
  bin_inp_f.read(reinterpret_cast<char *>(&i_f),sizeof(i_f));
  bin_inp_f.read(reinterpret_cast<char *>(&t),sizeof(t));
  bin_inp_f.read(reinterpret_cast<char *>(hpt),N*sizeof(ptype));
  bin_inp_f.read(reinterpret_cast<char *>(hr),N*sizeof(float4));
  bin_inp_f.read(reinterpret_cast<char *>(hlr),n_l*sizeof(float4));

  //copy host arrays to device
  cuda_check(hipMemcpy(pt,hpt,N*sizeof(ptype),hipMemcpyHostToDevice));
  cuda_check(hipMemcpy(r,hr,N*sizeof(float4),hipMemcpyHostToDevice));
  cuda_check(hipMemcpy(lr,hlr,n_l*sizeof(float4),hipMemcpyHostToDevice));

  // ----------------------------------------------------------------------------
  lg.generate_arrays(tpb,lr);

  //record success message
  logger::record("simulation checkpoint loaded");
}

//run simulation and write trajectory to binary file
void chrsim::run_simulation(std::ofstream &bin_out_f) //binary output file
{
  //iterate over all frames per file
  for (uint ffi = 0; ffi<fpf; ++ffi) //file frame index
  {
    //show simulation progress
    logger::show_prog_pc(100.0*ffi/fpf);

    //iterate over all steps per frame
    for (uint fsi = 0; fsi<spf; ++fsi) //frame step index
    {
      //make one Runge-Kutta iteration
      pg.generate_arrays(tpb,r);
      exec_RK_1<DST><<<(N+tpb-1)/tpb,tpb>>>
      (N,R,eps,pt,r,f,lr,er,sd,rn,vps,pgp,lgp);
      pg.generate_arrays(tpb,er);
      exec_RK_2<DST><<<(N+tpb-1)/tpb,tpb>>>
      (N,R,eps,pt,r,f,lr,er,ef,rn,pgp,lgp);
    }

    //copy position array to host
    cuda_check(hipMemcpy(hr,r,N*sizeof(float4),hipMemcpyDeviceToHost));

    //write trajectory frame
    ++i_f; t += spf*dt;
    write_frame_bin(bin_out_f);
  }

  //record success message
  logger::record("simulation ended");
}

//set random lbs positions
void chrsim::set_lbs_positions()
{
  //initialize host PRNG
  hiprandGenerator_t gen; //host PRNG
  hiprandCreateGeneratorHost(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen,time(nullptr));

  //declare auxiliary variables
  float ran; //random number in (0,1]
  float theta; //polar angle
  float phi; //azimuthal angle
  float3 ran_dir; //random direction

  //set lbs positions randomly
  for (uint i_l = 0; i_l<n_l; ++i_l) //lbs index
  {
    //generate random direction
    hiprandGenerateUniform(gen,&ran,1); theta = acos(1.0-2.0*ran);
    hiprandGenerateUniform(gen,&ran,1); phi = 2.0*M_PI*ran;
    ran_dir = {sin(theta)*cos(phi),sin(theta)*sin(phi),cos(theta)};

    //calculate position of next lbs
    hlr[i_l] = (R-1.0)*make_float4(ran_dir);

    //check if position is acceptable
    bool p_a = true; //position is acceptable
    for (uint j_l = 0; j_l<i_l; ++j_l) //secondary lbs index
    {
      float dll; //lbs lbs distance
      dll = length(make_float3(hlr[j_l]-hlr[i_l]));
      if (dll<1.0){ p_a = false;}
    }

    if (!p_a){ --i_l;} //repeat
  }

  //copy host lbs position array to device
  cuda_check(hipMemcpy(lr,hlr,n_l*sizeof(float4),hipMemcpyHostToDevice));

  // ----------------------------------------------------------------------------
  lg.generate_arrays(tpb,lr);

  //free host PRNG
  hiprandDestroyGenerator(gen);
}

//set random particle types
void chrsim::set_particle_types()
{
  //initialize host PRNG
  hiprandGenerator_t gen; //host PRNG
  hiprandCreateGeneratorHost(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen,time(nullptr));

  //set particle types randomly
  float ran; //random number in (0,1]
  for (uint i_p = 0; i_p<N; ++i_p) //particle index
  {
    hiprandGenerateUniform(gen,&ran,1);
    if (ran<0.5){ hpt[i_p] = LAD;}
    else{ hpt[i_p] = LND;}
  }

  //copy host particle type array to device
  cuda_check(hipMemcpy(pt,hpt,N*sizeof(ptype),hipMemcpyHostToDevice));

  //free host PRNG
  hiprandDestroyGenerator(gen);
}

//perform a confined random walk
void chrsim::perform_random_walk()
{
  //initialize host PRNG
  hiprandGenerator_t gen; //host PRNG
  hiprandCreateGeneratorHost(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen,time(nullptr));

  //declare auxiliary variables
  float iT = 1.0/(k_B*T); //inverse temperature
  float ran; //random number in (0,1]
  float theta; //polar angle
  float phi; //azimuthal angle
  float len_b; //bond length
  float angle_b; //bond angle
  float3 old_dir; //old direction
  float3 new_dir; //new direction
  float3 ran_dir; //random direction
  float3 per_dir; //perpendicular direction

  //place first particle
  hr[0] = {0.0,0.0,0.0,0.0};
  hiprandGenerateUniform(gen,&ran,1); theta = acos(1.0-2.0*ran);
  hiprandGenerateUniform(gen,&ran,1); phi = 2.0*M_PI*ran;
  ran_dir = {sin(theta)*cos(phi),sin(theta)*sin(phi),cos(theta)};
  old_dir = ran_dir;

  //place the rest of particles
  uint att = 0; //number of attempts
  for (uint i_p = 1; i_p<N; ++i_p) //particle index
  {
    //generate random direction perpendicular to old direction
    hiprandGenerateUniform(gen,&ran,1); theta = acos(1.0-2.0*ran);
    hiprandGenerateUniform(gen,&ran,1); phi = 2.0*M_PI*ran;
    ran_dir = {sin(theta)*cos(phi),sin(theta)*sin(phi),cos(theta)};
    per_dir = cross(old_dir,ran_dir);
    per_dir = normalize(per_dir);

    //generate random bond angle and calculate new direction
    hiprandGenerateUniform(gen,&ran,1);
    angle_b = acos(1.0+log(1.0-(1.0-exp(-2.0*(k_b*iT)))*ran)/(k_b*iT));
    new_dir = cos(angle_b)*old_dir+sin(angle_b)*per_dir;

    //calculate position of next particle
    hiprandGenerateUniform(gen,&ran,1);
    len_b = l_0+sqrt(2.0/(k_e*iT))*erfinv(2.0*ran-1.0);
    hr[i_p] = make_float4(len_b*new_dir)+hr[i_p-1];

    //check if new position is acceptable
    bool p_a = true; //position is acceptable
    if (!isfinite(hr[i_p].x)){ p_a = false;}
    if (!isfinite(hr[i_p].y)){ p_a = false;}
    if (!isfinite(hr[i_p].z)){ p_a = false;}
    float d_r; //radial distance to origin
    d_r = length(make_float3(hr[i_p]));
    if ((R-d_r)<mis){ p_a = false;}

    if (p_a) //continue
    {
      att = 0;
      old_dir = new_dir;
    }
    else //go back
    {
      ++att;
      if (att>1024){ i_p = 1+i_p*3/4;}
      else{ --i_p;}
    }
  }

  //copy host position array to device
  cuda_check(hipMemcpy(r,hr,N*sizeof(float4),hipMemcpyHostToDevice));

  //free host PRNG
  hiprandDestroyGenerator(gen);
}

//count particle overlaps
uint chrsim::particle_overlaps()
{
  //iterate over all pairs of non-bonded particles
  int po = 0; //particle overlaps
  for (uint i_p = 0; i_p<N; ++i_p) //particle index
  {
    for (uint j_p = 0; (j_p+1)<i_p; ++j_p) //secondary particle index
    {
      //check if particles overlap
      float dpp; //particle-particle distance
      dpp = length(make_float3(hr[j_p]-hr[i_p]));
      if (dpp<mis){ ++po;}
    }
  }
  return po;
}

} //namespace mmc
