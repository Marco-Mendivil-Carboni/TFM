#include "hip/hip_runtime.h"
//Includes

#include "chrsim.cuh"
#include "util.hpp"

#include <iostream> //standard input/output stream objects

#include <glob.h> //pathname pattern matching types

//Functions

int main(const int argc, const char **argv)
{
  //check command-line arguments
  if (argc<2){ std::cout<<"no arguments"<<std::endl; return EXIT_FAILURE;}
  if (argc>3){ std::cout<<"extra arguments"<<std::endl; return EXIT_FAILURE;}

  //declare auxiliary variables
  const std::string sim_dir = argv[1]; //simulation directory
  std::ifstream f_par; //parameter file
  std::ofstream f_out; //output file
  std::string f_path; //file path string
  int sim_idx = 0; //simulation index
  int tpf_idx = 0; //trajectory positions file index
  float t = 0.0; //simulation time ---------------- move to simulation class member variable ---------------------------

  //open log file inside sim_dir
  f_path = sim_dir+"/complete-history.log";
  mmcc::logger::set_file(f_path);

  try
  {
    //read parameters and initialize simulation
    f_path = sim_dir+"/adjustable-parameters.dat";
    f_par.open(f_path); mmcc::check_file(f_par,f_path);
    mmcc::chrsim sim(f_par);
    f_par.close();

    if (argc==2) //begin new simulation
    {
      glob_t prev_sims;
      std::string pattern = sim_dir+"/initial-configuration-*";
      if (glob(pattern.c_str(),0,nullptr,&prev_sims)==0)
      {
        sim_idx = prev_sims.gl_pathc;
      }
      globfree(&prev_sims); // move all this to some function
      mmcc::logger::record("new simulation started");

      sim.generate_initial_configuration();

      f_path = sim_dir+"/initial-configuration-";
      f_path += mmcc::cnfs(sim_idx,3)+".gro";
      f_out.open(f_path); mmcc::check_file(f_out,f_path);
      sim.write_initial_configuration(f_out);
      f_out.close();
    }
    else //continue previous simulation
    {
      sim_idx = std::stoi(argv[2]);
    }
    // log: sim_idx tpf_idx ?

    //perform simulation
    f_path = sim_dir+"/trajectory-positions-";
    f_path += mmcc::cnfs(sim_idx,3)+"-"+mmcc::cnfs(tpf_idx,3)+".trr";
    f_out.open(f_path,std::ios::binary); mmcc::check_file(f_out,f_path);
    //simulation
    f_out.close();
  }
  catch (const mmcc::error& error)
  {
    //exit program unsuccessfully
    mmcc::logger::record(error.what());
    return EXIT_FAILURE;
  }

  //exit program successfully
  return EXIT_SUCCESS;
}
