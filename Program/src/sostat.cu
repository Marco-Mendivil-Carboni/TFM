#include "hip/hip_runtime.h"
//Includes

#include "sostat.cuh" //simulation observable statistics

//Namespace

namespace mmc //Marco Mendívil Carboni
{

//Functions

//calculate last individual simulation statistics
void simobs::calc_last_is_stat()
{
  tdstat tas; //time series auxiliary statistics
  calc_stats(is_ts,tas);
  is_sv.push_back(tas);
}

//save last individual simulation statistics
void simobs::save_last_is_stat(std::ofstream &txt_out_f) //text output file
{
  tdstat tas = is_sv.back(); //time series auxiliary statistics
  txt_out_f<<cnfs(tas.avg,12,' ',6)<<cnfs(sqrt(tas.var),12,' ',6);
  txt_out_f<<cnfs(tas.sem,12,' ',6)<<cnfs(tas.f_n_b,8,' ');
  txt_out_f<<cnfs(tas.i_t,8,' ')<<(tas.thm?" yes":"  no")<<"\n";
}

//save last individual simulation statistics summary
void simobs::save_last_is_stat_s(std::ofstream &txt_out_f) //text output file
{
  tdstat tas = is_sv.back(); //time series auxiliary statistics
  txt_out_f<<cnfs(tas.avg,12,' ',6)<<cnfs(sqrt(tas.var),12,' ',6);
  txt_out_f<<cnfs(tas.sem,12,' ',6)<<"\n";
}

//calculate combined simulations final statistics
void simobs::calc_cs_final_stat()
{
  calc_stats(is_sv,cs_fs);
}

//save combined simulations final statistics
void simobs::save_cs_final_stat(std::ofstream &txt_out_f) //text output file
{
  txt_out_f<<cnfs(cs_fs.avg,12,' ',6)<<cnfs(sqrt(cs_fs.var),12,' ',6);
  txt_out_f<<cnfs(cs_fs.sem,12,' ',6)<<"\n";
}

//calculate statistics
void calc_stats(
  const std::vector<float> &v, //vector
  idstat &s) //statistics
{
  //calculate the first two raw moments
  double m_1 = 0.0; //1st moment
  double m_2 = 0.0; //2nd moment
  uint n_e = v.size(); //number of elements
  for (uint i_e = 0; i_e<n_e; ++i_e) //element index
  {
    m_1 += v[i_e];
    m_2 += v[i_e]*v[i_e];
  }
  m_1 /= n_e;
  m_2 /= n_e;

  //calculate statistics
  s.avg = m_1;
  s.var = (m_2-m_1*m_1)/(1.0-1.0/n_e);
  s.sem = sqrt(s.var/n_e);
}

//calculate statistics
void calc_stats(
  const std::vector<float> &v, //vector
  cdstat &s) //statistics
{
  //declare auxiliary variables
  std::vector<float> av = v; //auxiliary vector
  idstat ias; //independent auxiliary statistics

  //calculate average and variance
  calc_stats(av,ias);
  s.avg = ias.avg;
  s.var = ias.var;

  //calculate the standard error of the mean (by the blocking method)
  uint n_e = av.size(); //number of elements
  double ivm = ias.var/n_e; //independent variance of the mean
  double ulivm = ivm*(1.0+sqrt(2.0/(n_e-1.0))); //ivm upper limit
  while (n_e>=4)
  {
    //block data
    uint i_b; //block index
    for(i_b = 0; (2*i_b+1)<n_e; ++i_b)
    {
      av[i_b] = 0.5*(av[2*i_b]+av[2*i_b+1]);
    }
    n_e = i_b; av.resize(n_e);

    //calculate sem and f_n_b
    calc_stats(av,ias);
    ivm = ias.var/n_e;
    s.sem = sqrt(ivm);
    s.f_n_b = n_e;

    if (ivm>ulivm) //update the ivm upper limit
    {
      ulivm = ivm*(1.0+sqrt(2.0/(n_e-1.0)));
    }
    else //stop as the method has converged
    {
      break;
    }
  }
}

//calculate statistics
void calc_stats(
  const std::vector<float> &v, //vector
  tdstat &s) //statistics
{
  //declare auxiliary variables
  std::vector<float> av; //auxiliary vector
  idstat ias; //independent auxiliary statistics
  cdstat cas; //correlated auxiliary statistics

  //estimate thermalization (by the marginal standard error rule)
  double mse; //marginal standard error
  double min_mse = INFINITY; //minimum mse
  for(uint d = 2; d<128; d*=2) //denominator
  {
    //remove thermalization vector elements
    uint i_t = v.size()/d; //thermalization index
    av = {v.begin()+i_t,v.end()};
    uint n_e = av.size(); //number of elements

    //calculate the marginal standard error
    calc_stats(av,ias);
    mse = ias.var*(n_e-1)/(n_e*n_e);

    //save the optimal thermalization index
    if (mse<min_mse)
    {
      min_mse = mse;
      s.i_t = i_t;
    }
  }

  //determine if data has thermalized
  if (s.i_t!=v.size()/2){ s.thm = true;} //thermalized
  else{ s.thm = false;} //did not thermalize

  //calculate the rest of the statistics
  av = {v.begin()+s.i_t,v.end()};
  calc_stats(av,cas);
  s.avg = cas.avg;
  s.var = cas.var;
  s.sem = cas.sem;
  s.f_n_b = cas.f_n_b;
}

//calculate statistics
void calc_stats(
  const std::vector<tdstat> &v, //vector
  idstat &s) //statistics
{
  //calculate the first two weighted raw moments
  double m_1 = 0.0; //1st moment
  double m_2 = 0.0; //2nd moment
  double w_1 = 0.0; //1st weight sum
  double w_2 = 0.0; //2nd weight sum
  uint n_e = v.size(); //number of elements
  for (uint i_e = 0; i_e<n_e; ++i_e) //element index
  {
    double w = 1.0/(v[i_e].sem*v[i_e].sem); //weight
    if (!isfinite(w)){ w = 1.0;}
    m_1 += w*v[i_e].avg;
    m_2 += w*v[i_e].avg*v[i_e].avg;
    w_1 += w;
    w_2 += w*w;
  }
  m_1 /= w_1;
  m_2 /= w_1;

  //calculate weighted statistics
  s.avg = m_1;
  s.var = (m_2-m_1*m_1)/(1.0-w_2/(w_1*w_1));
  s.sem = sqrt(s.var*w_2/(w_1*w_1));
}

} //namespace mmc
