#include "hip/hip_runtime.h"
//Includes

#include "sugrid.cuh" //chromatin simulation

#include "util.cuh" //general utilities
#include "vecops.cuh" //vector operations

#include <cub/device/device_radix_sort.cuh> //cub parallel radix sort

//Namespace

namespace mmc //Marco Mendívil Carboni
{

//Global Functions

//calculate grid cell and particle indexes
__global__ void calc_indexes(
  const int n_o, //number of objects
  const float csl, //grid cell side length
  const uint cps, //grid cells per side
  uint *uci, //unsorted grid cell index array
  uint *upi, //unsorted particle index array
  float4 *r) //position array
{
  //calculate particle index
  int i_p = blockIdx.x*blockDim.x+threadIdx.x; //particle index
  if (i_p>=n_o){ return;}
  upi[i_p] = i_p;

  //calculate auxiliary variables
  float3 r_i = make_float3(r[i_p]); //particle position
  int3 ir = floorf(r_i/csl); //integer coordinates
  int iofst = (cps/2)*(1+cps+cps*cps); //index offset

  //calculate grid cell index
  uci[i_p] = iofst+ir.x+ir.y*cps+ir.z*cps*cps;
}

//set grid cells empty
__global__ void set_cells_empty(
  const uint gc, //number of grid cells
  sugrid *gp) //grid pointer
{
  //calculate array index
  int i_a = blockIdx.x*blockDim.x+threadIdx.x; //array index
  if (i_a>=gc){ return;}

  //set beginning and end of grid cells
  gp->beg[i_a] = 0xffffffff; gp->end[i_a] = 0;
}

//find beginning and end of each grid cell
__global__ void find_cells_limits(
  const int N, //number of particles
  float4 *r, //position array
  sugrid *gp) //grid pointer
{
  //calculate array index
  int i_a = blockIdx.x*blockDim.x+threadIdx.x; //array index
  if (i_a>=N){ return;}

  //set beginning and end of cells
  int ci_curr = gp->sci[i_a]; //current cell index
  if (i_a==0)
  {
    gp->beg[ci_curr] = i_a; return;
  }
  int ci_prev = gp->sci[i_a-1]; //previous cell index
  if (ci_prev!=ci_curr)
  {
    gp->beg[ci_curr] = i_a;
    gp->end[ci_prev] = i_a;
  }
  if (i_a==N-1)
  {
    gp->end[ci_curr] = i_a+1;
  }
}

//Host Functions

//sorted uniform grid constructor
sugrid::sugrid(
    const uint n_o, //number of objects
    const float csl, //grid cell side length
    const uint cps) //grid cells per side
  : n_o {n_o}
  , csl {csl}
  , cps {cps}
  , n_c {cps*cps*cps}
{
  //check parameters
  if (csl<0.0){ throw error("grid_cell_side_length out of range");}
  if (cps<1){ throw error("grid_cells_per_side out of range");}
  std::string msg = "sugrid:"; //message
  msg += " csl = "+cnfs(csl,6,'0',2);
  msg += " cps = "+cnfs(cps,5,'0');
  logger::record(msg);

  //allocate arrays
  cuda_check(hipMalloc(&uci,n_o*sizeof(uint)));
  cuda_check(hipMalloc(&sci,n_o*sizeof(uint)));
  cuda_check(hipMalloc(&upi,n_o*sizeof(uint)));
  cuda_check(hipMalloc(&spi,n_o*sizeof(uint)));
  cuda_check(hipMalloc(&beg,n_c*sizeof(uint)));
  cuda_check(hipMalloc(&end,n_c*sizeof(uint)));
  cuda_check(hipMalloc(&sr,n_o*sizeof(float4)));

  //allocate extra buffer
  hipcub::DeviceRadixSort::SortPairs(nullptr,ebs,uci,sci,upi,spi,n_o);
  cuda_check(hipMalloc(&eb,ebs));
}

//sorted uniform grid destructor
sugrid::~sugrid()
{
  //deallocate arrays
  cuda_check(hipFree(uci));
  cuda_check(hipFree(sci));
  cuda_check(hipFree(upi));
  cuda_check(hipFree(spi));
  cuda_check(hipFree(beg));
  cuda_check(hipFree(end));
  cuda_check(hipFree(sr));

  //deallocate extra buffer
  cuda_check(hipFree(eb));
}

//generate grid lists
void sugrid::generate_lists(
  int tpb, //threads per block
  float4 *r) //position array
{
  calc_indexes<<<(n_o+tpb-1)/tpb,tpb>>>(n_o,csl,cps,uci,upi,r);
  hipcub::DeviceRadixSort::SortPairs(eb,ebs,uci,sci,upi,spi,n_o);
  set_cells_empty<<<(n_c+tpb-1)/tpb,tpb>>>(ljc,ljp);
  find_cells_limits<<<(n_o+tpb-1)/tpb,tpb>>>(N,r,ljp);
}

} //namespace mmc
