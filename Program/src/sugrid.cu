#include "hip/hip_runtime.h"
//Includes

#include "sugrid.cuh" //chromatin simulation

#include "util.cuh" //general utilities
#include "vecops.cuh" //vector operations

#include <cub/device/device_radix_sort.cuh> //cub parallel radix sort

//Namespace

namespace mmc //Marco Mendívil Carboni
{

//Global Functions

//calculate cell and particle indexes
__global__ void calc_indexes(
  const uint n_o, //number of objects
  const float csl, //cell side length
  const uint cps, //cells per side
  uint *uci, //unsorted cell index array
  uint *upi, //unsorted particle index array
  float4 *r) //position array
{
  //calculate particle index
  int i_p = blockIdx.x*blockDim.x+threadIdx.x; //particle index
  if (i_p>=n_o){ return;}
  upi[i_p] = i_p;

  //calculate auxiliary variables
  float3 r_i = make_float3(r[i_p]); //particle position
  int3 ir = floorf(r_i/csl); //integer coordinates
  int iofst = (cps/2)*(1+cps+cps*cps); //index offset

  //calculate cell index
  uci[i_p] = iofst+ir.x+ir.y*cps+ir.z*cps*cps;
}

//set cells empty
__global__ void set_cells_empty(
  const uint n_c, //number of cells
  uint *beg, //cell beginning array
  uint *end) //cell end array
{
  //calculate limit array index
  int lai = blockIdx.x*blockDim.x+threadIdx.x; //limit array index
  if (lai>=n_c){ return;}

  //set beginning and end of cells
  beg[lai] = 0;
  end[lai] = 0;
}

//find beginning and end of each cell
__global__ void find_cells_limits(
  const uint n_o, //number of objects
  uint *sci, //sorted cell index array
  uint *beg, //cell beginning array
  uint *end) //cell end array
{
  //calculate sorted array index
  int sai = blockIdx.x*blockDim.x+threadIdx.x; //sorted array index
  if (sai>=n_o){ return;}

  //set beginning and end of cells
  int ci_curr = sci[sai]; //current cell index
  if (sai==0)
  {
    beg[ci_curr] = sai;
    return;
  }
  int ci_prev = sci[sai-1]; //previous cell index
  if (ci_prev!=ci_curr)
  {
    beg[ci_curr] = sai;
    end[ci_prev] = sai;
  }
  if (sai==n_o-1)
  {
    end[ci_curr] = sai+1;
  }
}

//Host Functions

//sorted uniform grid constructor
sugrid::sugrid(
    const uint n_o, //number of objects
    const float csl, //cell side length
    const uint cps) //cells per side
  : n_o {n_o}
  , csl {csl}
  , cps {cps}
  , n_c {cps*cps*cps}
{
  //allocate arrays
  cuda_check(hipMalloc(&uci,n_o*sizeof(uint)));
  cuda_check(hipMalloc(&sci,n_o*sizeof(uint)));
  cuda_check(hipMalloc(&upi,n_o*sizeof(uint)));
  cuda_check(hipMalloc(&spi,n_o*sizeof(uint)));
  cuda_check(hipMalloc(&beg,n_c*sizeof(uint)));
  cuda_check(hipMalloc(&end,n_c*sizeof(uint)));

  //allocate extra buffer
  hipcub::DeviceRadixSort::SortPairs(nullptr,ebs,uci,sci,upi,spi,n_o);
  cuda_check(hipMalloc(&eb,ebs));
}

//sorted uniform grid delegating constructor
sugrid::sugrid(
    const uint n_o, //number of objects
    const sugrid &g) //grid
  : sugrid(n_o,g.csl,g.cps) {}

//sorted uniform grid destructor
sugrid::~sugrid()
{
  //deallocate arrays
  cuda_check(hipFree(uci));
  cuda_check(hipFree(sci));
  cuda_check(hipFree(upi));
  cuda_check(hipFree(spi));
  cuda_check(hipFree(beg));
  cuda_check(hipFree(end));

  //deallocate extra buffer
  cuda_check(hipFree(eb));
}

//generate grid arrays
void sugrid::generate_arrays(
  int tpb, //threads per block
  float4 *r) //position array
{
  calc_indexes<<<(n_o+tpb-1)/tpb,tpb>>>(n_o,csl,cps,uci,upi,r);
  hipcub::DeviceRadixSort::SortPairs(eb,ebs,uci,sci,upi,spi,n_o);
  set_cells_empty<<<(n_c+tpb-1)/tpb,tpb>>>(n_c,beg,end);
  find_cells_limits<<<(n_o+tpb-1)/tpb,tpb>>>(n_o,sci,beg,end);
}

} //namespace mmc
