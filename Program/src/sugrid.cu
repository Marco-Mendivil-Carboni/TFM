#include "hip/hip_runtime.h"
// Includes

#include "sugrid.cuh" //chromatin simulation

#include <cub/device/device_radix_sort.cuh> //cub parallel radix sort

// Namespace

namespace mmc // Marco Mendívil Carboni
{

// Global Functions

// calculate cell and object indexes
__global__ void calc_indexes(const uint n_o, // number of objects
    const float csl, // cell side length
    const uint cps, // cells per side
    uint *uci, // unsorted cell index array
    uint *uoi, // unsorted object index array
    vec3f *r) // position array
{
  // calculate object index
  int i_o = blockIdx.x * blockDim.x + threadIdx.x; // object index
  if (i_o >= n_o)
  {
    return;
  }
  uoi[i_o] = i_o;

  // calculate auxiliary variables
  vec3f r_i = r[i_o]; // object position
  vec3i ir = ifloorc(r_i / csl); // integer coordinates
  int iofst = (cps / 2) * (1 + cps + cps * cps); // index offset

  // calculate cell index
  uci[i_o] = iofst + ir.x + ir.y * cps + ir.z * cps * cps;
}

// set cells empty
__global__ void set_cells_empty(const uint n_c, // number of cells
    uint *beg, // cell beginning array
    uint *end) // cell end array
{
  // calculate limit array index
  int lai = blockIdx.x * blockDim.x + threadIdx.x; // limit array index
  if (lai >= n_c)
  {
    return;
  }

  // set beginning and end of cells
  beg[lai] = 0;
  end[lai] = 0;
}

// find beginning and end of each cell
__global__ void find_cells_limits(const uint n_o, // number of objects
    uint *sci, // sorted cell index array
    uint *beg, // cell beginning array
    uint *end) // cell end array
{
  // calculate sorted array index
  int sai = blockIdx.x * blockDim.x + threadIdx.x; // sorted array index
  if (sai >= n_o)
  {
    return;
  }

  // set beginning and end of cells
  int ci_curr = sci[sai]; // current cell index
  if (sai == 0)
  {
    beg[ci_curr] = sai;
    return;
  }
  int ci_prev = sci[sai - 1]; // previous cell index
  if (ci_prev != ci_curr)
  {
    beg[ci_curr] = sai;
    end[ci_prev] = sai;
  }
  if (sai == n_o - 1)
  {
    end[ci_curr] = sai + 1;
  }
}

// Host Functions

// sorted uniform grid constructor
sugrid::sugrid(const uint n_o, // number of objects
    const float csl, // cell side length
    const uint cps) // cells per side
    : n_o{n_o}, csl{csl}, cps{cps}, n_c{cps * cps * cps}
{
  // allocate arrays
  cuda_check(hipMalloc(&uci, n_o * sizeof(uint)));
  cuda_check(hipMalloc(&sci, n_o * sizeof(uint)));
  cuda_check(hipMalloc(&uoi, n_o * sizeof(uint)));
  cuda_check(hipMalloc(&soi, n_o * sizeof(uint)));
  cuda_check(hipMalloc(&beg, n_c * sizeof(uint)));
  cuda_check(hipMalloc(&end, n_c * sizeof(uint)));

  // allocate extra buffer
  hipcub::DeviceRadixSort::SortPairs(nullptr, ebs, uci, sci, uoi, soi, n_o);
  cuda_check(hipMalloc(&eb, ebs));
}

// sorted uniform grid delegating constructor
sugrid::sugrid(const uint n_o, // number of objects
    const sugrid &g) // grid
    : sugrid(n_o, g.csl, g.cps)
{
}

// sorted uniform grid destructor
sugrid::~sugrid()
{
  // deallocate arrays
  cuda_check(hipFree(uci));
  cuda_check(hipFree(sci));
  cuda_check(hipFree(uoi));
  cuda_check(hipFree(soi));
  cuda_check(hipFree(beg));
  cuda_check(hipFree(end));

  // deallocate extra buffer
  cuda_check(hipFree(eb));
}

// generate grid arrays
void sugrid::generate_arrays(int tpb, // threads per block
    vec3f *r) // position array
{
  calc_indexes<<<(n_o + tpb - 1) / tpb, tpb>>>(n_o, csl, cps, uci, uoi, r);
  hipcub::DeviceRadixSort::SortPairs(eb, ebs, uci, sci, uoi, soi, n_o);
  set_cells_empty<<<(n_c + tpb - 1) / tpb, tpb>>>(n_c, beg, end);
  find_cells_limits<<<(n_o + tpb - 1) / tpb, tpb>>>(n_o, sci, beg, end);
}

} // namespace mmc
